#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <cmath>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/axes.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/data_type.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/cuda/numeric.cuh"
#include "chainerx/cuda/numeric_limits.cuh"
#include "chainerx/cuda/reduce.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/kernels/math.h"
#include "chainerx/kernels/sorting.h"
#include "chainerx/macro.h"
#include "chainerx/numeric_limits.h"
#include "chainerx/reduction_kernel_arg.h"
#include "chainerx/routines/math.h"
#include "chainerx/shape.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct ArgMaxImpl {
    using CudaType = cuda_internal::DataType<T>;
    struct MaxAndArgMax {
        CudaType max;
        int64_t argmax;
    };
    __device__ MaxAndArgMax Identity() { return {CudaType{}, -1}; }
    __device__ MaxAndArgMax MapIn(CudaType in, int64_t index) { return {in, index}; }
    __device__ void Reduce(MaxAndArgMax next, MaxAndArgMax& accum) {
        // Note that `next` can be the return value of `Identity()` in which case `accum` should not be updated.
        if (next.argmax != -1 && (accum.argmax == -1 || accum.max < next.max)) {
            accum = next;
        }
    }
    __device__ int64_t MapOut(MaxAndArgMax accum) { return accum.argmax; }
};

class CudaArgMaxKernel : public ArgMaxKernel {
public:
    void Call(const Array& a, const Axes& axis, const Array& out) override {
        Device& device = a.device();
        device.CheckDevicesCompatible(a, out);
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(a.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Reduce<T, int64_t>(a, axis, out, ArgMaxImpl<T>{});
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(ArgMaxKernel, CudaArgMaxKernel);

template <typename In, typename Out>
struct SumImpl {
    using InCudaType = cuda_internal::DataType<In>;
    using OutCudaType = cuda_internal::DataType<Out>;
    __device__ OutCudaType Identity() { return OutCudaType{0}; }
    __device__ OutCudaType MapIn(InCudaType in, int64_t /*index*/) { return static_cast<OutCudaType>(in); }
    __device__ void Reduce(OutCudaType next, OutCudaType& accum) { accum += next; }
    __device__ OutCudaType MapOut(OutCudaType accum) { return accum; }
};

class CudaSumKernel : public SumKernel {
public:
    void Call(const Array& a, const Axes& axis, const Array& out) override {
        Device& device = a.device();
        CHAINERX_ASSERT(internal::IsValidReductionShape(a.shape(), axis, out.shape(), true));
        device.CheckDevicesCompatible(a, out);
        CudaSetDeviceScope scope{device.index()};

        auto do_sum = [&a, &axis, &out](auto in_pt, auto out_pt) {
            using In = typename decltype(in_pt)::type;
            using Out = typename decltype(out_pt)::type;
            Reduce<In, Out>(a, axis, out, SumImpl<In, Out>{});
        };

        VisitDtype(out.dtype(), [a_dtype = a.dtype(), &do_sum](auto out_pt) { VisitDtype(a_dtype, do_sum, out_pt); });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(SumKernel, CudaSumKernel);

template <typename T>
struct AMaxImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ CudaType Identity() { return cuda::NumericLimits<CudaType>::LowestOrInf(); }
    __device__ CudaType MapIn(CudaType in, int64_t /*index*/) { return in; }
    __device__ void Reduce(CudaType next, CudaType& accum) {
        if (cuda::IsNan(next) || accum < next) {
            accum = next;
        }
    }
    __device__ CudaType MapOut(CudaType accum) { return accum; }
};

class CudaAMaxKernel : public AMaxKernel {
public:
    void Call(const Array& a, const Axes& axis, const Array& out) override {
        Device& device = a.device();
        CHAINERX_ASSERT(internal::IsValidReductionShape(a.shape(), axis, out.shape(), true));
        device.CheckDevicesCompatible(a, out);
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Reduce<T, T>(a, axis, out, AMaxImpl<T>{});
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(AMaxKernel, CudaAMaxKernel);

template <typename T>
struct AMinImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ CudaType Identity() { return cuda::NumericLimits<CudaType>::MaxOrInf(); }
    __device__ CudaType MapIn(CudaType in, int64_t /*index*/) { return in; }
    __device__ void Reduce(CudaType next, CudaType& accum) {
        if (cuda::IsNan(next) || accum > next) {
            accum = next;
        }
    }
    __device__ CudaType MapOut(CudaType accum) { return accum; }
};

class CudaAMinKernel : public AMinKernel {
public:
    void Call(const Array& a, const Axes& axis, const Array& out) override {
        Device& device = a.device();
        CHAINERX_ASSERT(internal::IsValidReductionShape(a.shape(), axis, out.shape(), true));
        device.CheckDevicesCompatible(a, out);
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Reduce<T, T>(a, axis, out, AMinImpl<T>{});
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(AMinKernel, CudaAMinKernel);

}  // namespace
}  // namespace cuda
}  // namespace chainerx
