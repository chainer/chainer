#include "chainerx/cuda/cuda_device.h"

#include <cmath>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/cuda/numeric.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/kernels/math.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct ExpImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = cuda::Exp(x); }
};

class CudaExpKernel : public ExpKernel {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
        VisitFloatingPointDtype(out.dtype(), [&x_cast, &out](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, T>(ExpImpl<T>{}, x_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(ExpKernel, CudaExpKernel);

template <typename T>
struct LogImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = cuda::Log(x); }
};

class CudaLogKernel : public LogKernel {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
        VisitFloatingPointDtype(out.dtype(), [&x_cast, &out](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, T>(LogImpl<T>{}, x_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(LogKernel, CudaLogKernel);

}  // namespace
}  // namespace cuda
}  // namespace chainerx
