#include "chainerx/cuda/cuda_device.h"

#include <cmath>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/numeric.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct ExpImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = cuda::Exp(x); }
};

}  // namespace

void CudaDevice::Exp(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CudaSetDeviceScope scope{index()};
    const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
    VisitFloatingPointDtype(out.dtype(), [&x_cast, &out](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, T>(ExpImpl<T>{}, x_cast, out);
    });
}

namespace {

template <typename T>
struct LogImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = cuda::Log(x); }
};

}  // namespace

void CudaDevice::Log(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CudaSetDeviceScope scope{index()};
    const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
    VisitFloatingPointDtype(out.dtype(), [&x_cast, &out](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, T>(LogImpl<T>{}, x_cast, out);
    });
}

}  // namespace cuda
}  // namespace chainerx
