#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <cmath>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct ExpImpl {
    __device__ void operator()(int64_t /*i*/, T x, T& out) { out = std::exp(x); }
};

}  // namespace

void CudaDevice::Exp(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CudaSetDeviceScope scope{index()};
    VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, T>(ExpImpl<T>{}, x, out);
    });
}

namespace {

template <typename T>
struct LogImpl {
    __device__ void operator()(int64_t /*i*/, T x, T& out) { out = std::log(x); }
};

}  // namespace

void CudaDevice::Log(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CudaSetDeviceScope scope{index()};
    VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, T>(LogImpl<T>{}, x, out);
    });
}

}  // namespace cuda
}  // namespace chainerx
