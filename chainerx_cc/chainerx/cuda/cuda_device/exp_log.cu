#include "chainerx/cuda/cuda_device.h"

#include <cmath>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/numeric.cuh"
#include "chainerx/cuda/op_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/routines/math.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct ExpImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = cuda::Exp(x); }
};

class CudaExpOp : public ExpOp {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
        VisitFloatingPointDtype(out.dtype(), [&x_cast, &out](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, T>(ExpImpl<T>{}, x_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_OP(ExpOp, CudaExpOp);

template <typename T>
struct LogImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = cuda::Log(x); }
};

class CudaLogOp : public LogOp {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
        VisitFloatingPointDtype(out.dtype(), [&x_cast, &out](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, T>(LogImpl<T>{}, x_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_OP(LogOp, CudaLogOp);

}  // namespace
}  // namespace cuda
}  // namespace chainerx
