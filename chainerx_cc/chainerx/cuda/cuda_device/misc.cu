#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <cmath>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct SqrtImpl {
    __device__ void operator()(int64_t /*i*/, T x, T& out) { out = std::sqrt(x); }
};

}  // namespace

void CudaDevice::Sqrt(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CudaSetDeviceScope scope{index()};
    VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, T>(SqrtImpl<T>{}, x, out);
    });
}

namespace {

template <typename T>
__device__ bool IsNan(T /*value*/) {
    return false;
}
__device__ bool IsNan(double value) { return isnan(value); }
__device__ bool IsNan(float value) { return isnan(value); }

template <typename T>
struct IsNanImpl {
    __device__ void operator()(int64_t /*i*/, T x, bool& out) { out = IsNan(x); }
};

}  // namespace

void CudaDevice::IsNan(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(x.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, bool>(IsNanImpl<T>{}, x, out);
    });
}

namespace {

template <typename T>
__device__ bool IsInf(T /*value*/) {
    return false;
}
__device__ bool IsInf(double value) { return isinf(value); }
__device__ bool IsInf(float value) { return isinf(value); }

template <typename T>
struct IsInfImpl {
    __device__ void operator()(int64_t /*i*/, T x, bool& out) { out = IsInf(x); }
};

}  // namespace

void CudaDevice::IsInf(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(x.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, bool>(IsInfImpl<T>{}, x, out);
    });
}

}  // namespace cuda
}  // namespace chainerx
