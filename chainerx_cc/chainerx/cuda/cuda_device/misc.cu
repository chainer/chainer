#include "chainerx/cuda/cuda_device.h"

#include <cmath>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/cuda/numeric.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/kernels/math.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct SquareImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = x * x; }
};

class CudaSquareKernel : public SquareKernel {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, T>(SquareImpl<T>{}, x, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(SquareKernel, CudaSquareKernel);

template <typename T>
struct SqrtImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = cuda::Sqrt(x); }
};

class CudaSqrtKernel : public SqrtKernel {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, T>(SqrtImpl<T>{}, x_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(SqrtKernel, CudaSqrtKernel);

template <typename T>
struct IsNanImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, bool& out) { out = cuda::IsNan(x); }
};

class CudaIsNanKernel : public IsNanKernel {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(x.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, bool>(IsNanImpl<T>{}, x, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(IsNanKernel, CudaIsNanKernel);

template <typename T>
struct IsInfImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, bool& out) { out = cuda::IsInf(x); }
};

class CudaIsInfKernel : public IsInfKernel {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(x.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, bool>(IsInfImpl<T>{}, x, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(IsInfKernel, CudaIsInfKernel);

template <typename T>
struct CeilImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = cuda::Ceil(x); }
};

class CudaCeilKernel : public CeilKernel {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
        VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, T>(CeilImpl<T>{}, x_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(CeilKernel, CudaCeilKernel);

template <typename T>
struct FloorImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = cuda::Floor(x); }
};

class CudaFloorKernel : public FloorKernel {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
        VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, T>(FloorImpl<T>{}, x_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(FloorKernel, CudaFloorKernel);

}  // namespace
}  // namespace cuda
}  // namespace chainerx
