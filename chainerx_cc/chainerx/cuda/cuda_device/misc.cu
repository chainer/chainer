#include "chainerx/cuda/cuda_device.h"

#include <cmath>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/numeric.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct SqrtImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = cuda::Sqrt(x); }
};

}  // namespace

void CudaDevice::Sqrt(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CudaSetDeviceScope scope{index()};
    VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, T>(SqrtImpl<T>{}, x, out);
    });
}

namespace {

template <typename T>
struct PowImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, CudaType& out) { out = cuda::Pow(x1, x2); }
};

}  // namespace

void CudaDevice::Pow(const Array& x1, const Array& x2, const Array& out) {
    CheckDevicesCompatible(x1, x2, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, const T, T>(PowImpl<T>{}, x1, x2, out);
    });
}

namespace {

template <typename T>
struct PowASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = cuda::Pow(x1, x2); }
    CudaType x2;
};

}  // namespace

void CudaDevice::PowAS(const Array& x1, Scalar x2, const Array& out) {
    CheckDevicesCompatible(x1, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        using CudaType = cuda_internal::DataType<T>;
        Elementwise<const T, T>(PowASImpl<T>{static_cast<CudaType>(x2)}, x1, out);
    });
}

namespace {

    template <typename T>
    struct PowSAImpl {
        using CudaType = cuda_internal::DataType<T>;
        __device__ void operator()(int64_t /*i*/, CudaType x2, CudaType& out) { out = cuda::Pow(x1, x2); }
        CudaType x1;
    };
    
}  // namespace
    
void CudaDevice::PowSA(Scalar x1,const Array& x2,const Array& out){
    CheckDevicesCompatible(x2, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        using CudaType = cuda_internal::DataType<T>;
        Elementwise<const T,T>(PowSAImpl<T>{static_cast<CudaType>(x1)}, x2, out);
    });
}
    

namespace {

template <typename T>
struct IsNanImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, bool& out) { out = cuda::IsNan(x); }
};

}  // namespace

void CudaDevice::IsNan(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(x.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, bool>(IsNanImpl<T>{}, x, out);
    });
}

namespace {

template <typename T>
struct IsInfImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, bool& out) { out = cuda::IsInf(x); }
};

}  // namespace

void CudaDevice::IsInf(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(x.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, bool>(IsInfImpl<T>{}, x, out);
    });
}

}  // namespace cuda
}  // namespace chainerx
