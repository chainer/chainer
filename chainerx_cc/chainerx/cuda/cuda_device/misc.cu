#include "chainerx/cuda/cuda_device.h"

#include <cmath>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/cuda/numeric.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/kernels/misc.h"
#include "chainerx/routines/type_util.h"

namespace chainerx {
namespace cuda {
namespace {

CHAINERX_CUDA_REGISTER_ELTWISE_FLOAT_UNARY_KERNEL(SqrtKernel, { out = cuda::Sqrt(x); });

CHAINERX_CUDA_REGISTER_ELTWISE_DTYPE_UNARY_KERNEL(SquareKernel, { out = x * x; }, VisitNumericDtype);

CHAINERX_CUDA_REGISTER_ELTWISE_DTYPE_UNARY_KERNEL(AbsKernel, { out = cuda::Abs(x); }, VisitNumericDtype);

CHAINERX_CUDA_REGISTER_ELTWISE_DTYPE_UNARY_KERNEL(SignKernel, { out = cuda::Sign(x); }, VisitNumericDtype);

template <typename In, typename Out>
struct IfLessElseASSAImpl {
    using InCudaType = cuda_internal::DataType<In>;
    using OutCudaType = cuda_internal::DataType<Out>;
    __device__ void operator()(int64_t /*i*/, InCudaType x1, OutCudaType neg, OutCudaType& out) { out = x1 < x2 ? pos : neg; }
    InCudaType x2;
    OutCudaType pos;
};

class CudaIfLessElseASSAKernel : public IfLessElseASSAKernel {
public:
    void Call(const Array& x1, Scalar x2, Scalar pos, const Array& neg, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, neg, out);
        Dtype x_dtype = ResultType(x1, x2);
        const Array& x1_cast = x1.dtype() == x_dtype ? x1 : x1.AsType(x_dtype);
        const Array& neg_cast = neg.dtype() == out.dtype() ? neg : neg.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(x_dtype, [&](auto x_pt) {
            using In = typename decltype(x_pt)::type;
            using InCudaType = cuda_internal::DataType<In>;
            VisitNumericDtype(out.dtype(), [&](auto pt) {
                using Out = typename decltype(pt)::type;
                using OutCudaType = cuda_internal::DataType<Out>;
                Elementwise<const In, const Out, Out>(
                        IfLessElseASSAImpl<In, Out>{static_cast<InCudaType>(x2), static_cast<OutCudaType>(pos)}, x1_cast, neg_cast, out);
            });
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(IfLessElseASSAKernel, CudaIfLessElseASSAKernel);

template <typename In, typename Out>
struct IfGreaterElseASSAImpl {
    using InCudaType = cuda_internal::DataType<In>;
    using OutCudaType = cuda_internal::DataType<Out>;
    __device__ void operator()(int64_t /*i*/, InCudaType x1, OutCudaType neg, OutCudaType& out) { out = x1 > x2 ? pos : neg; }
    InCudaType x2;
    OutCudaType pos;
};

class CudaIfGreaterElseASSAKernel : public IfGreaterElseASSAKernel {
public:
    void Call(const Array& x1, Scalar x2, Scalar pos, const Array& neg, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, neg, out);
        Dtype x_dtype = ResultType(x1, x2);
        const Array& x1_cast = x1.dtype() == x_dtype ? x1 : x1.AsType(x_dtype);
        const Array& neg_cast = neg.dtype() == out.dtype() ? neg : neg.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(x_dtype, [&](auto x_pt) {
            using In = typename decltype(x_pt)::type;
            using InCudaType = cuda_internal::DataType<In>;
            VisitNumericDtype(out.dtype(), [&](auto pt) {
                using Out = typename decltype(pt)::type;
                using OutCudaType = cuda_internal::DataType<Out>;
                Elementwise<const In, const Out, Out>(
                        IfGreaterElseASSAImpl<In, Out>{static_cast<InCudaType>(x2), static_cast<OutCudaType>(pos)}, x1_cast, neg_cast, out);
            });
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(IfGreaterElseASSAKernel, CudaIfGreaterElseASSAKernel);

template <typename In, typename Out>
struct IfGreaterElseAAAAImpl {
    using InCudaType = cuda_internal::DataType<In>;
    using OutCudaType = cuda_internal::DataType<Out>;
    __device__ void operator()(int64_t /*i*/, InCudaType x1, InCudaType x2, OutCudaType pos, OutCudaType neg, OutCudaType& out) {
        out = x1 > x2 ? pos : neg;
    }
};

class CudaIfGreaterElseAAAAKernel : public IfGreaterElseAAAAKernel {
public:
    void Call(const Array& x1, const Array& x2, const Array& pos, const Array& neg, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, x2, pos, neg, out);
        Dtype x_dtype = ResultType(x1, x2);
        const Array& x1_cast = x1.dtype() == x_dtype ? x1 : x1.AsType(x_dtype);
        const Array& x2_cast = x2.dtype() == x_dtype ? x2 : x2.AsType(x_dtype);
        const Array& pos_cast = pos.dtype() == out.dtype() ? pos : pos.AsType(out.dtype());
        const Array& neg_cast = neg.dtype() == out.dtype() ? neg : neg.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(x_dtype, [&](auto x_pt) {
            using In = typename decltype(x_pt)::type;
            VisitNumericDtype(out.dtype(), [&](auto pt) {
                using Out = typename decltype(pt)::type;
                Elementwise<const In, const In, const Out, const Out, Out>(
                        IfGreaterElseAAAAImpl<In, Out>{}, x1_cast, x2_cast, pos_cast, neg_cast, out);
            });
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(IfGreaterElseAAAAKernel, CudaIfGreaterElseAAAAKernel);

}  // namespace
}  // namespace cuda
}  // namespace chainerx
