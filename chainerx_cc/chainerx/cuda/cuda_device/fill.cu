#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <algorithm>
#include <cstdint>
#include <mutex>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/cast.cuh"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/dtype.h"
#include "chainerx/indexable_array.h"
#include "chainerx/indexer.h"
#include "chainerx/macro.h"
#include "chainerx/scalar.h"
#include "chainerx/shape.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct ArangeImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t i, CudaType& out) { out = start + step * static_cast<CudaType>(i); }
    CudaType start;
    CudaType step;
};

}  // namespace

void CudaDevice::Arange(Scalar start, Scalar step, const Array& out) {
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        using CudaType = cuda_internal::DataType<T>;
        Elementwise<T>(ArangeImpl<T>{static_cast<CudaType>(start), static_cast<CudaType>(step)}, out);
    });
}

namespace {

template <typename T>
struct FillImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType& out) { out = value; }
    CudaType value;
};

}  // namespace

void CudaDevice::Fill(const Array& out, Scalar value) {
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        using CudaType = cuda_internal::DataType<T>;
        Elementwise<T>(FillImpl<T>{static_cast<CudaType>(value)}, out);
    });
}

namespace {

template <typename T>
struct IdentityImpl {
    using CudaType = cuda_internal::DataType<T>;
    explicit IdentityImpl(int64_t n) : n_plus_one{n + 1} {}
    __device__ void operator()(int64_t i, CudaType& out) { out = i % n_plus_one == 0 ? CudaType{1} : CudaType{0}; }
    int64_t n_plus_one;
};

}  // namespace

void CudaDevice::Identity(const Array& out) {
    CHAINERX_ASSERT(out.ndim() == 2);
    CHAINERX_ASSERT(out.shape()[0] == out.shape()[1]);

    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<T>(IdentityImpl<T>{out.shape()[0]}, out);
    });
}

namespace {

template <typename T>
struct EyeImpl {
    using CudaType = cuda_internal::DataType<T>;
    EyeImpl(int64_t m, int64_t k) : start{k < 0 ? -k * m : k}, stop{m * (m - k)}, step{m + 1} {}
    __device__ void operator()(int64_t i, CudaType& out) {
        out = start <= i && i < stop && (i - start) % step == 0 ? CudaType{1} : CudaType{0};
    }
    int64_t start;
    int64_t stop;
    int64_t step;
};

}  // namespace

void CudaDevice::Eye(int64_t k, const Array& out) {
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [k, &out](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<T>(EyeImpl<T>{out.shape()[1], k}, out);
    });
}

namespace {

template <typename T>
__global__ void SetVecInMat(
        IndexableArray<const T, 1> vec_iarray,
        IndexableArray<T, 2> mat_iarray,
        Indexer<1> vec_indexer,
        Indexer<2> mat_indexer,
        int64_t mat_row_start,
        int64_t mat_col_start) {
    auto mat_it = mat_indexer.It(0);
    for (auto vec_it = vec_indexer.It(blockIdx.x * blockDim.x + threadIdx.x, blockDim.x * gridDim.x); vec_it; ++vec_it) {
        mat_it.index()[0] = mat_row_start + vec_it.raw_index();
        mat_it.index()[1] = mat_col_start + vec_it.raw_index();
        mat_iarray[mat_it] = vec_iarray[vec_it];
    }
}

}  // namespace

void CudaDevice::Diagflat(const Array& v, int64_t k, const Array& out) {
    CHAINERX_ASSERT(v.ndim() == 1);
    CHAINERX_ASSERT(out.ndim() == 2);

    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;

        // Start indices for the 2-D array axes with applied offset k.
        int64_t row_start{0};
        int64_t col_start{0};

        if (k >= 0) {
            col_start += k;
        } else {
            row_start -= k;
        }

        // Initialize all elements to 0 first instead of conditionally filling in the diagonal.
        Fill(out, T{0});

        IndexableArray<const T, 1> v_iarray{v};
        IndexableArray<T, 2> out_iarray{out};
        Indexer<1> v_indexer{v.shape()};
        Indexer<2> out_indexer{out.shape()};

        // TODO(niboshi): Calculate kMaxBlockSize per device
        std::lock_guard<std::mutex> lock{*cuda_internal::g_mutex};
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&SetVecInMat<T>).block_size;
        int64_t total_size = out_indexer.total_size();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        SetVecInMat<<<grid_size, block_size>>>(v_iarray, out_iarray, v_indexer, out_indexer, row_start, col_start);
    });
}

namespace {

template <typename T>
struct LinspaceImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t i, CudaType& out) {
        double value = n == 1 ? start : (start * (n - 1 - i) + stop * i) / (n - 1);
        out = cuda_numeric_cast<CudaType>(value);
    }
    int64_t n;
    double start;
    double stop;
};

}  // namespace

void CudaDevice::Linspace(double start, double stop, const Array& out) {
    CHAINERX_ASSERT(out.ndim() == 1);
    CHAINERX_ASSERT(out.shape()[0] > 0);

    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        int64_t n = out.shape()[0];
        Elementwise<T>(LinspaceImpl<T>{n, start, stop}, out);
    });
}

}  // namespace cuda
}  // namespace chainerx
