#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <algorithm>
#include <cstdint>
#include <mutex>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/cast.cuh"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/indexable_array.h"
#include "chainerx/indexer.h"
#include "chainerx/kernels/creation.h"
#include "chainerx/kernels/misc.h"
#include "chainerx/macro.h"
#include "chainerx/routines/creation.h"
#include "chainerx/scalar.h"
#include "chainerx/shape.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct ArangeImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t i, CudaType& out) { out = start + step * static_cast<CudaType>(i); }
    CudaType start;
    CudaType step;
};

class CudaArangeKernel : public ArangeKernel {
public:
    void Call(Scalar start, Scalar step, const Array& out) override {
        Device& device = out.device();
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<T>(ArangeImpl<T>{static_cast<CudaType>(start), static_cast<CudaType>(step)}, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(ArangeKernel, CudaArangeKernel);

template <typename T>
struct IdentityImpl {
    using CudaType = cuda_internal::DataType<T>;
    explicit IdentityImpl(int64_t n) : n_plus_one{n + 1} {}
    __device__ void operator()(int64_t i, CudaType& out) { out = i % n_plus_one == 0 ? CudaType{1} : CudaType{0}; }
    int64_t n_plus_one;
};

class CudaIdentityKernel : public IdentityKernel {
public:
    void Call(const Array& out) override {
        CHAINERX_ASSERT(out.ndim() == 2);
        CHAINERX_ASSERT(out.shape()[0] == out.shape()[1]);

        Device& device = out.device();
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<T>(IdentityImpl<T>{out.shape()[0]}, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(IdentityKernel, CudaIdentityKernel);

template <typename T>
struct EyeImpl {
    using CudaType = cuda_internal::DataType<T>;
    EyeImpl(int64_t m, int64_t k) : start{k < 0 ? -k * m : k}, stop{m * (m - k)}, step{m + 1} {}
    __device__ void operator()(int64_t i, CudaType& out) {
        out = start <= i && i < stop && (i - start) % step == 0 ? CudaType{1} : CudaType{0};
    }
    int64_t start;
    int64_t stop;
    int64_t step;
};

class CudaEyeKernel : public EyeKernel {
public:
    void Call(int64_t k, const Array& out) override {
        Device& device = out.device();
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [k, &out](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<T>(EyeImpl<T>{out.shape()[1], k}, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(EyeKernel, CudaEyeKernel);

template <typename T>
__global__ void SetVecInMat(
        IndexableArray<const T, 1> vec_iarray,
        IndexableArray<T, 2> mat_iarray,
        Indexer<1> vec_indexer,
        Indexer<2> mat_indexer,
        int64_t mat_row_start,
        int64_t mat_col_start) {
    auto mat_it = mat_indexer.It(0);
    for (auto vec_it = vec_indexer.It(blockIdx.x * blockDim.x + threadIdx.x, blockDim.x * gridDim.x); vec_it; ++vec_it) {
        mat_it.index()[0] = mat_row_start + vec_it.raw_index();
        mat_it.index()[1] = mat_col_start + vec_it.raw_index();
        mat_iarray[mat_it] = vec_iarray[vec_it];
    }
}

class CudaDiagflatKernel : public DiagflatKernel {
public:
    void Call(const Array& v, int64_t k, const Array& out) override {
        CHAINERX_ASSERT(v.ndim() == 1);
        CHAINERX_ASSERT(out.ndim() == 2);

        Device& device = v.device();
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;

            // Start indices for the 2-D array axes with applied offset k.
            int64_t row_start{0};
            int64_t col_start{0};

            if (k >= 0) {
                col_start += k;
            } else {
                row_start -= k;
            }

            // Initialize all elements to 0 first instead of conditionally filling in the diagonal.
            device.backend().CallKernel<FillKernel>(out, T{0});

            IndexableArray<const T, 1> v_iarray{v};
            IndexableArray<T, 2> out_iarray{out};
            Indexer<1> v_indexer{v.shape()};
            Indexer<2> out_indexer{out.shape()};

            // TODO(niboshi): Calculate kMaxBlockSize per device
            std::lock_guard<std::mutex> lock{*cuda_internal::g_mutex};
            static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&SetVecInMat<T>).block_size;
            int64_t total_size = out_indexer.total_size();
            int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
            int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

            SetVecInMat<<<grid_size, block_size>>>(v_iarray, out_iarray, v_indexer, out_indexer, row_start, col_start);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(DiagflatKernel, CudaDiagflatKernel);

template <typename T>
struct LinspaceImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t i, CudaType& out) {
        double value = n == 1 ? start : (start * (n - 1 - i) + stop * i) / (n - 1);
        out = cuda_numeric_cast<CudaType>(value);
    }
    int64_t n;
    double start;
    double stop;
};

class CudaLinspaceKernel : public LinspaceKernel {
public:
    void Call(double start, double stop, const Array& out) override {
        CHAINERX_ASSERT(out.ndim() == 1);
        CHAINERX_ASSERT(out.shape()[0] > 0);

        Device& device = out.device();
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            int64_t n = out.shape()[0];
            Elementwise<T>(LinspaceImpl<T>{n, start, stop}, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(LinspaceKernel, CudaLinspaceKernel);

template <typename T>
struct FillImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType& out) { out = value; }
    CudaType value;
};

class CudaFillKernel : public FillKernel {
public:
    void Call(const Array& out, Scalar value) override {
        CudaSetDeviceScope scope{out.device().index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<T>(FillImpl<T>{static_cast<CudaType>(value)}, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(FillKernel, CudaFillKernel);

template <typename T>
struct TriImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t i, CudaType& out) {
        int64_t row = i / m;
        int64_t col = i % m;
        out = col <= row + k ? CudaType{1} : CudaType{0};
    }
    int64_t m;
    int64_t k;
};

class CudaTriKernel : public TriKernel {
public:
    void Call(int64_t k, const Array& out) override {
        Device& device = out.device();
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [k, &out](auto pt) {
            using T = typename decltype(pt)::type;
            int64_t m = out.shape()[1];
            Elementwise<T>(TriImpl<T>{m, k}, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(TriKernel, CudaTriKernel);

}  // namespace
}  // namespace cuda
}  // namespace chainerx
