#include "chainerx/cuda/cuda_device.h"

#include <cmath>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/numeric.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/numeric.h"
#include "chainerx/scalar.h"

namespace chainerx {
namespace cuda {

namespace {

template <typename T>
struct IfLessElseASSAImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType neg, CudaType& out) { out = x1 < x2 ? pos : neg; }
    CudaType x2;
    CudaType pos;
};

}  // namespace

void CudaDevice::IfLessElseASSA(const Array& x1, Scalar x2, Scalar pos, const Array& neg, const Array& out) {
    CheckDevicesCompatible(x1, neg, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        using CudaType = cuda_internal::DataType<T>;
        Elementwise<const T, const T, T>(IfLessElseASSAImpl<T>{static_cast<CudaType>(x2), static_cast<CudaType>(pos)}, x1, neg, out);
    });
}

namespace {

template <typename T>
struct IfGreaterElseASSAImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType neg, CudaType& out) { out = x1 > x2 ? pos : neg; }
    CudaType x2;
    CudaType pos;
};

}  // namespace

void CudaDevice::IfGreaterElseASSA(const Array& x1, Scalar x2, Scalar pos, const Array& neg, const Array& out) {
    CheckDevicesCompatible(x1, neg, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        using CudaType = cuda_internal::DataType<T>;
        Elementwise<const T, const T, T>(IfGreaterElseASSAImpl<T>{static_cast<CudaType>(x2), static_cast<CudaType>(pos)}, x1, neg, out);
    });
}

namespace {

template <typename T>
struct TanhImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = cuda::Tanh(x); }
};

}  // namespace

void CudaDevice::Tanh(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CudaSetDeviceScope scope{index()};
    const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
    VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, T>(TanhImpl<T>{}, x_cast, out);
    });
}

}  // namespace cuda
}  // namespace chainerx
