#include "chainerx/cuda/cuda_device.h"

#include <cstdint>
#include <mutex>
#include <type_traits>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cuda_fp16.hpp>

#include "chainerx/array.h"
#include "chainerx/axes.h"
#include "chainerx/backend.h"
#include "chainerx/backend_util.h"
#include "chainerx/cuda/hipblas.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/data_type.cuh"
#include "chainerx/cuda/float16.cuh"
#include "chainerx/cuda/op_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/error.h"
#include "chainerx/float16.h"
#include "chainerx/macro.h"
#include "chainerx/routines/creation.h"
#include "chainerx/routines/linalg.h"
#include "chainerx/routines/math.h"
#include "chainerx/routines/misc.h"

namespace chainerx {
namespace cuda {
namespace {

// Dispatch gemm routines based on the element type T
template <typename T>
struct Gemm;

template <>
struct Gemm<float> {
    template <typename... Args>
    hipblasStatus_t operator()(Args&&... args) const {
        return hipblasSgemm(std::forward<Args>(args)...);
    }
};

template <>
struct Gemm<double> {
    template <typename... Args>
    hipblasStatus_t operator()(Args&&... args) const {
        return hipblasDgemm(std::forward<Args>(args)...);
    }
};

struct GemmInputLayout {
    int64_t ld = 0;
    hipblasOperation_t trans = HIPBLAS_OP_T;

    // Configure leading dimension and transposition accordingly, and makes the array C contiguous if necessary.
    Array Configure(const Array& a) {
        CHAINERX_ASSERT(a.ndim() == 2);
        // Row-major
        // Note that this condition is slightly relaxed than Array::IsContiguous() which requires
        // a.strides()[0] == a.GetItemSize() * a.shape()[1]
        if (a.strides()[1] == a.GetItemSize() && a.strides()[0] / a.GetItemSize() >= a.shape()[1] &&
            a.strides()[0] % a.GetItemSize() == 0) {
            ld = a.strides()[0] / a.GetItemSize();
            trans = HIPBLAS_OP_N;  // transposed
            return a;
        }
        // Column-major
        if (a.strides()[0] == a.GetItemSize() && a.strides()[1] / a.GetItemSize() >= a.shape()[0] &&
            a.strides()[1] % a.GetItemSize() == 0) {
            ld = a.strides()[1] / a.GetItemSize();
            return a;
        }
        // Force row-major contiguous
        ld = a.shape()[1];
        trans = HIPBLAS_OP_N;  // transposed
        return internal::AsContiguous(a);
    }
};

}  // namespace

class CudaDotOp : public DotOp {
public:
    void Call(const Array& a, const Array& b, const Array& out) override {
        Device& device = a.device();
        device.CheckDevicesCompatible(a, b, out);
        CudaSetDeviceScope scope{device.index()};

        if (GetKind(out.dtype()) != DtypeKind::kFloat) {
            throw NotImplementedError("dot is not implemented for non-float types in CUDA");
        }

        CHAINERX_ASSERT(a.ndim() == 2);
        CHAINERX_ASSERT(b.ndim() == 2);
        CHAINERX_ASSERT(out.ndim() == 2);

        int64_t m = a.shape()[0];
        int64_t k = a.shape()[1];
        int64_t n = b.shape()[1];
        CHAINERX_ASSERT(b.shape()[0] == k);
        CHAINERX_ASSERT(out.shape()[0] == m);
        CHAINERX_ASSERT(out.shape()[1] == n);

        if (m == 1 && n == 1) {
            // TODO(beam2d): Write a custom reduction kernel.
            // TODO(hvy): Avoid unnecessary cast here when multiplication supports mixed dtypes.
            const Array& a_cast = a.dtype() == out.dtype() ? a : a.AsType(out.dtype());
            const Array& b_cast = b.dtype() == out.dtype() ? b : b.AsType(out.dtype());
            device.backend().CallOp<SumOp>(a_cast.Reshape({k}) * b_cast.Reshape({k}), Axes{0}, out.Reshape({}));
            return;
        }

        if (out.dtype() == Dtype::kFloat16) {
            // TODO(imanishi): Use hipblasHgemm
            Array out_float32 = Empty(out.shape(), Dtype::kFloat32, device);
            device.backend().CallOp<DotOp>(a.AsType(Dtype::kFloat32), b.AsType(Dtype::kFloat32), out_float32);
            device.backend().CallOp<AsTypeOp>(out_float32, out);
            return;
        }

        bool is_out_contiguous = out.IsContiguous();
        Array out_contiguous = is_out_contiguous ? out : EmptyLike(out, device);

        const Array& a_cast = a.dtype() == out.dtype() ? a : a.AsType(out.dtype());
        const Array& b_cast = b.dtype() == out.dtype() ? b : b.AsType(out.dtype());

        auto gemm_impl = [&](auto pt) {
            CHAINERX_ASSERT(a_cast.dtype() == out_contiguous.dtype());
            CHAINERX_ASSERT(b_cast.dtype() == out_contiguous.dtype());

            using T = typename decltype(pt)::type;
            using StorageType = cuda_internal::StorageType<T>;
            using CudaType = cuda_internal::DataType<T>;

            // Note that cuBLAS uses Fortran order.
            // To compute out = a x b, we use cuBLAS to compute out^T = b^T x a^T (here x is the matrix product).

            GemmInputLayout a_cast_layout;
            GemmInputLayout b_cast_layout;
            Array a_cast_config = a_cast_layout.Configure(a_cast);
            Array b_cast_config = b_cast_layout.Configure(b_cast);

            const CudaType one{chainerx::Float16{1}};
            const CudaType zero{chainerx::Float16{0}};
            const CudaType* a_cast_ptr =
                    &cuda_internal::StorageToDataType<const T>(*static_cast<const StorageType*>(internal::GetRawOffsetData(a_cast_config)));
            const CudaType* b_cast_ptr =
                    &cuda_internal::StorageToDataType<const T>(*static_cast<const StorageType*>(internal::GetRawOffsetData(b_cast_config)));
            CudaType* out_ptr =
                    &cuda_internal::StorageToDataType<T>(*static_cast<StorageType*>(internal::GetRawOffsetData(out_contiguous)));

            cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(static_cast<CudaDevice&>(device));

            device_internals.cublas_handle().Call(
                    Gemm<T>{},
                    b_cast_layout.trans,
                    a_cast_layout.trans,
                    n,
                    m,
                    k,
                    &one,
                    b_cast_ptr,
                    b_cast_layout.ld,
                    a_cast_ptr,
                    a_cast_layout.ld,
                    &zero,
                    out_ptr,
                    n);
        };

        switch (out.dtype()) {
            case Dtype::kFloat32:
                gemm_impl(PrimitiveType<float>{});
                break;
            case Dtype::kFloat64:
                gemm_impl(PrimitiveType<double>{});
                break;
            default:
                CHAINERX_NEVER_REACH();
        }

        if (!is_out_contiguous) {
            device.backend().CallOp<CopyOp>(out_contiguous, out);
        }
    }
};

CHAINERX_CUDA_REGISTER_OP(DotOp, CudaDotOp);

}  // namespace cuda
}  // namespace chainerx
