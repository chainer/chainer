#include "chainerx/cuda/cuda_device.h"

#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/kernels/creation.h"
#include "chainerx/kernels/misc.h"
#include "chainerx/routines/creation.h"

namespace chainerx {
namespace cuda {
namespace {

CHAINERX_CUDA_REGISTER_ELTWISE_UNARY_KERNEL(CopyKernel, { out = x; });

template <typename InT, typename OutT>
struct AsTypeImpl {
    using InCudaType = cuda_internal::DataType<InT>;
    using OutCudaType = cuda_internal::DataType<OutT>;
    __device__ void operator()(int64_t /*i*/, InCudaType a, OutCudaType& out) { out = static_cast<OutCudaType>(a); }
};

class CudaAsTypeKernel : public AsTypeKernel {
public:
    void Call(const Array& a, const Array& out) override {
        Device& device = a.device();
        device.CheckDevicesCompatible(a, out);
        CudaSetDeviceScope scope{device.index()};
        auto do_astype = [&](auto in_pt, auto out_pt) {
            using InT = typename decltype(in_pt)::type;
            using OutT = typename decltype(out_pt)::type;
            Elementwise<const InT, OutT>(AsTypeImpl<InT, OutT>{}, a, out);
        };
        VisitDtype(out.dtype(), [&](auto out_pt) { VisitDtype(a.dtype(), do_astype, out_pt); });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(AsTypeKernel, CudaAsTypeKernel);

}  // namespace
}  // namespace cuda
}  // namespace chainerx
