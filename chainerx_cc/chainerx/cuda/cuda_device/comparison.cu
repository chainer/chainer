#include "chainerx/cuda/cuda_device.h"

#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/data_type.cuh"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/op_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/routines/logic.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct EqualImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, bool& out) { out = x1 == x2; }
};

class CudaEqualOp : public EqualOp {
protected:
    void Impl(const Array& x1, const Array& x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, x2, out);
        Dtype dtype = PromoteTypes(x1.dtype(), x2.dtype());
        const Array& x1_cast = x1.dtype() == dtype ? x1 : x1.AsType(dtype);
        const Array& x2_cast = x2.dtype() == dtype ? x2 : x2.AsType(dtype);
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(dtype, [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, const T, bool>(EqualImpl<T>{}, x1_cast, x2_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(EqualOp, CudaEqualOp);

template <typename T>
struct NotEqualImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, bool& out) { out = x1 != x2; }
};

class CudaNotEqualOp : public NotEqualOp {
protected:
    void Impl(const Array& x1, const Array& x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, x2, out);
        Dtype dtype = PromoteTypes(x1.dtype(), x2.dtype());
        const Array& x1_cast = x1.dtype() == dtype ? x1 : x1.AsType(dtype);
        const Array& x2_cast = x2.dtype() == dtype ? x2 : x2.AsType(dtype);
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(dtype, [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, const T, bool>(NotEqualImpl<T>{}, x1_cast, x2_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(NotEqualOp, CudaNotEqualOp);

template <typename T>
struct GreaterImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, bool& out) { out = x1 > x2; }
};

class CudaGreaterOp : public GreaterOp {
protected:
    void Impl(const Array& x1, const Array& x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, x2, out);
        Dtype dtype = PromoteTypes(x1.dtype(), x2.dtype());
        const Array& x1_cast = x1.dtype() == dtype ? x1 : x1.AsType(dtype);
        const Array& x2_cast = x2.dtype() == dtype ? x2 : x2.AsType(dtype);
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(dtype, [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, const T, bool>(GreaterImpl<T>{}, x1_cast, x2_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(GreaterOp, CudaGreaterOp);

template <typename T>
struct GreaterEqualImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, bool& out) { out = x1 >= x2; }
};

class CudaGreaterEqualOp : public GreaterEqualOp {
protected:
    void Impl(const Array& x1, const Array& x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, x2, out);
        Dtype dtype = PromoteTypes(x1.dtype(), x2.dtype());
        const Array& x1_cast = x1.dtype() == dtype ? x1 : x1.AsType(dtype);
        const Array& x2_cast = x2.dtype() == dtype ? x2 : x2.AsType(dtype);
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(dtype, [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, const T, bool>(GreaterEqualImpl<T>{}, x1_cast, x2_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(GreaterEqualOp, CudaGreaterEqualOp);

template <typename T>
struct LogicalNotImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, bool& out) { out = !x; }
};

class CudaLogicalNotOp : public LogicalNotOp {
protected:
    void Impl(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(x.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, bool>(LogicalNotImpl<T>{}, x, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(LogicalNotOp, CudaLogicalNotOp);

}  // namespace
}  // namespace cuda
}  // namespace chainerx
