#include "chainerx/cuda/cuda_device.h"

#include <cmath>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/cuda/numeric.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/kernels/trigonometric.h"
#include "chainerx/numeric.h"
#include "chainerx/scalar.h"

namespace chainerx {
namespace cuda {
namespace {

CHAINERX_CUDA_REGISTER_ELTWISE_FLOAT_UNARY_KERNEL(SinKernel, { out = cuda::Sin(x); });

CHAINERX_CUDA_REGISTER_ELTWISE_FLOAT_UNARY_KERNEL(CosKernel, { out = cuda::Cos(x); });

CHAINERX_CUDA_REGISTER_ELTWISE_FLOAT_UNARY_KERNEL(TanKernel, { out = cuda::Tan(x); });

CHAINERX_CUDA_REGISTER_ELTWISE_FLOAT_UNARY_KERNEL(ArcsinKernel, { out = cuda::Arcsin(x); });

CHAINERX_CUDA_REGISTER_ELTWISE_FLOAT_UNARY_KERNEL(ArccosKernel, { out = cuda::Arccos(x); });

CHAINERX_CUDA_REGISTER_ELTWISE_FLOAT_UNARY_KERNEL(ArctanKernel, { out = cuda::Arctan(x); });

CHAINERX_CUDA_REGISTER_ELTWISE_FLOAT_BINARY_KERNEL(Arctan2Kernel, { out = cuda::Arctan2(x1, x2); });

}  // namespace
}  // namespace cuda
}  // namespace chainerx
