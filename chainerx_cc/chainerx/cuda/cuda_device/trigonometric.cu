#include "chainerx/cuda/cuda_device.h"

#include <cmath>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/numeric.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/numeric.h"
#include "chainerx/scalar.h"

namespace chainerx {
namespace cuda {

namespace {

template <typename T>
struct TanImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = cuda::Tan(x); }
};

}  // namespace

void CudaDevice::Tan(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CudaSetDeviceScope scope{index()};
    const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
    VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, T>(TanImpl<T>{}, x_cast, out);
    });
}

namespace {

template <typename T>
struct ArcsinImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = cuda::Arcsin(x); }
};

}  // namespace

void CudaDevice::Arcsin(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CudaSetDeviceScope scope{index()};
    const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
    VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, T>(ArcsinImpl<T>{}, x_cast, out);
    });
}

namespace {

template <typename T>
struct ArccosImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = cuda::Arccos(x); }
};

}  // namespace

void CudaDevice::Arccos(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CudaSetDeviceScope scope{index()};
    const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
    VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, T>(ArccosImpl<T>{}, x_cast, out);
    });
}

namespace {

template <typename T>
struct ArctanImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = cuda::Arctan(x); }
};

}  // namespace

void CudaDevice::Arctan(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CudaSetDeviceScope scope{index()};
    const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
    VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, T>(ArctanImpl<T>{}, x_cast, out);
    });
}

}  // namespace cuda
}  // namespace chainerx
