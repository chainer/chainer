#include "chainerx/cuda/cuda_device.h"

#include <cstdint>
#include <mutex>
#include <type_traits>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <cuda_fp16.hpp>

#include "chainerx/array.h"
#include "chainerx/axes.h"
#include "chainerx/backend.h"
#include "chainerx/backend_util.h"
#include "chainerx/cuda/hipblas.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/cusolver.h"
#include "chainerx/cuda/data_type.cuh"
#include "chainerx/cuda/float16.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/error.h"
#include "chainerx/float16.h"
#include "chainerx/kernels/creation.h"
#include "chainerx/kernels/linalg.h"
#include "chainerx/kernels/misc.h"
#include "chainerx/macro.h"
#include "chainerx/native/native_device.h"
#include "chainerx/routines/creation.h"
#include "chainerx/routines/linalg.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
hipsolverStatus_t GetrfBuffersize(hipsolverHandle_t /*handle*/, int /*m*/, int /*n*/, T* /*a*/, int /*lda*/, int* /*lwork*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by getrf (LU)"};
}

template <typename T>
hipsolverStatus_t Getrf(
        hipsolverHandle_t /*handle*/, int /*m*/, int /*n*/, T* /*a*/, int /*lda*/, T* /*workspace*/, int* /*devipiv*/, int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by getrf (LU)"};
}

template <typename T>
hipsolverStatus_t Getrs(
        hipsolverHandle_t /*handle*/,
        hipblasOperation_t /*trans*/,
        int /*n*/,
        int /*nrhs*/,
        T* /*a*/,
        int /*lda*/,
        int* /*devipiv*/,
        T* /*b*/,
        int /*ldb*/,
        int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by getrs (Solve)"};
}

template <>
hipsolverStatus_t GetrfBuffersize<double>(hipsolverHandle_t handle, int m, int n, double* a, int lda, int* lwork) {
    return hipsolverDnDgetrf_bufferSize(handle, m, n, a, lda, lwork);
}

template <>
hipsolverStatus_t GetrfBuffersize<float>(hipsolverHandle_t handle, int m, int n, float* a, int lda, int* lwork) {
    return hipsolverDnSgetrf_bufferSize(handle, m, n, a, lda, lwork);
}

template <>
hipsolverStatus_t Getrf<double>(hipsolverHandle_t handle, int m, int n, double* a, int lda, double* workspace, int* devipiv, int* devinfo) {
    return hipsolverDnDgetrf(handle, m, n, a, lda, workspace, devipiv, devinfo);
}

template <>
hipsolverStatus_t Getrf<float>(hipsolverHandle_t handle, int m, int n, float* a, int lda, float* workspace, int* devipiv, int* devinfo) {
    return hipsolverDnSgetrf(handle, m, n, a, lda, workspace, devipiv, devinfo);
}

template <>
hipsolverStatus_t Getrs<double>(
        hipsolverHandle_t handle,
        hipblasOperation_t trans,
        int n,
        int nrhs,
        double* a,
        int lda,
        int* devipiv,
        double* b,
        int ldb,
        int* devinfo) {
    return hipsolverDnDgetrs(handle, trans, n, nrhs, a, lda, devipiv, b, ldb, devinfo);
}

template <>
hipsolverStatus_t Getrs<float>(
        hipsolverHandle_t handle,
        hipblasOperation_t trans,
        int n,
        int nrhs,
        float* a,
        int lda,
        int* devipiv,
        float* b,
        int ldb,
        int* devinfo) {
    return hipsolverDnSgetrs(handle, trans, n, nrhs, a, lda, devipiv, b, ldb, devinfo);
}

template <typename T>
void SolveImpl(const Array& a, const Array& b, const Array& out) {
    Device& device = a.device();
    Dtype dtype = a.dtype();

    cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(static_cast<CudaDevice&>(device));

    Array lu_matrix = Empty(a.shape(), dtype, device);
    device.backend().CallKernel<CopyKernel>(a.Transpose(), lu_matrix);
    auto lu_ptr = static_cast<T*>(internal::GetRawOffsetData(lu_matrix));

    int64_t m = a.shape()[0];
    int64_t nrhs = 1;
    if (b.ndim() == 2) {
        nrhs = b.shape()[1];
    }

    Array ipiv = Empty(Shape{m}, Dtype::kInt32, device);
    auto ipiv_ptr = static_cast<int*>(internal::GetRawOffsetData(ipiv));

    int buffersize = 0;
    device_internals.cusolverdn_handle().Call(GetrfBuffersize<T>, m, m, lu_ptr, m, &buffersize);

    Array work = Empty(Shape{buffersize}, dtype, device);
    auto work_ptr = static_cast<T*>(internal::GetRawOffsetData(work));

    std::shared_ptr<void> devinfo = device.Allocate(sizeof(int));

    device_internals.cusolverdn_handle().Call(Getrf<T>, m, m, lu_ptr, m, work_ptr, ipiv_ptr, static_cast<int*>(devinfo.get()));

    int devinfo_h = 0;
    Device& native_device = GetDefaultContext().GetDevice({"native", 0});
    device.MemoryCopyTo(&devinfo_h, devinfo.get(), sizeof(int), native_device);
    if (devinfo_h != 0) {
        throw ChainerxError{"Unsuccessful getrf (LU) execution. Info = ", devinfo_h};
    }

    Array out_transposed = b.Transpose().Copy();
    auto out_ptr = static_cast<T*>(internal::GetRawOffsetData(out_transposed));

    device_internals.cusolverdn_handle().Call(
            Getrs<T>, HIPBLAS_OP_N, m, nrhs, lu_ptr, m, ipiv_ptr, out_ptr, m, static_cast<int*>(devinfo.get()));

    device.MemoryCopyTo(&devinfo_h, devinfo.get(), sizeof(int), native_device);
    if (devinfo_h != 0) {
        throw ChainerxError{"Unsuccessful getrs (Solve) execution. Info = ", devinfo_h};
    }

    device.backend().CallKernel<CopyKernel>(out_transposed.Transpose(), out);
}

}  // namespace

class CudaSolveKernel : public SolveKernel {
public:
    void Call(const Array& a, const Array& b, const Array& out) override {
        Device& device = a.device();
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);
        CHAINERX_ASSERT(a.shape()[0] == a.shape()[1]);

        VisitFloatingPointDtype(dtype, [&](auto pt) {
            using T = typename decltype(pt)::type;
            SolveImpl<T>(a, b, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(SolveKernel, CudaSolveKernel);

class CudaInverseKernel : public InverseKernel {
public:
    void Call(const Array& a, const Array& out) override {
        Device& device = a.device();
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);
        CHAINERX_ASSERT(a.shape()[0] == a.shape()[1]);

        // There is LAPACK routine ``getri`` for computing the inverse of an LU-factored matrix,
        // but cuSOLVER does not have it implemented, therefore inverse is obtained with ``getrs``
        // inv(A) == solve(A, Identity)
        Array b = Identity(a.shape()[0], dtype, device);
        device.backend().CallKernel<SolveKernel>(a, b, out);
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(InverseKernel, CudaInverseKernel);

}  // namespace cuda
}  // namespace chainerx
