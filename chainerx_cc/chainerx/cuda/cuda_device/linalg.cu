#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <cstdint>
#include <mutex>
#include <type_traits>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <cuda_fp16.hpp>

#include "chainerx/array.h"
#include "chainerx/axes.h"
#include "chainerx/backend.h"
#include "chainerx/backend_util.h"
#include "chainerx/cuda/hipblas.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/cusolver.h"
#include "chainerx/cuda/data_type.cuh"
#include "chainerx/cuda/float16.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/error.h"
#include "chainerx/float16.h"
#include "chainerx/kernels/creation.h"
#include "chainerx/kernels/linalg.h"
#include "chainerx/kernels/misc.h"
#include "chainerx/macro.h"
#include "chainerx/native/native_device.h"
#include "chainerx/routines/arithmetic.h"
#include "chainerx/routines/creation.h"
#include "chainerx/routines/indexing.h"
#include "chainerx/routines/linalg.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
hipsolverStatus_t GetrfBuffersize(hipsolverHandle_t /*handle*/, int /*m*/, int /*n*/, T* /*a*/, int /*lda*/, int* /*lwork*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by getrf (LU)"};
}

template <typename T>
hipsolverStatus_t Getrf(
        hipsolverHandle_t /*handle*/, int /*m*/, int /*n*/, T* /*a*/, int /*lda*/, T* /*workspace*/, int* /*devipiv*/, int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by getrf (LU)"};
}

template <typename T>
hipsolverStatus_t Getrs(
        hipsolverHandle_t /*handle*/,
        hipblasOperation_t /*trans*/,
        int /*n*/,
        int /*nrhs*/,
        T* /*a*/,
        int /*lda*/,
        int* /*devipiv*/,
        T* /*b*/,
        int /*ldb*/,
        int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by getrs (Solve)"};
}

template <typename T>
hipsolverStatus_t GesvdBuffersize(hipsolverHandle_t /*handle*/, int /*m*/, int /*n*/, int* /*lwork*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by gesvd (SVD)"};
}

template <typename T>
hipsolverStatus_t Gesvd(
        hipsolverHandle_t /*handle*/,
        signed char /*jobu*/,
        signed char /*jobvt*/,
        int /*m*/,
        int /*n*/,
        T* /*a*/,
        int /*lda*/,
        T* /*s*/,
        T* /*u*/,
        int /*ldu*/,
        T* /*vt*/,
        int /*ldvt*/,
        T* /*work*/,
        int /*lwork*/,
        T* /*rwork*/,
        int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by gesvd (SVD)"};
}

template <typename T>
hipsolverStatus_t GeqrfBufferSize(hipsolverHandle_t /*handle*/, int /*m*/, int /*n*/, T* /*a*/, int /*lda*/, int* /*lwork*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by geqrf (QR)"};
}

template <typename T>
hipsolverStatus_t Geqrf(
        hipsolverHandle_t /*handle*/,
        int /*m*/,
        int /*n*/,
        T* /*a*/,
        int /*lda*/,
        T* /*tau*/,
        T* /*workspace*/,
        int /*lwork*/,
        int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by geqrf (QR)"};
}

template <typename T>
hipsolverStatus_t OrgqrBufferSize(
        hipsolverHandle_t /*handle*/, int /*m*/, int /*n*/, int /*k*/, T* /*a*/, int /*lda*/, T* /*tau*/, int* /*lwork*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by orgqr (QR)"};
}

template <typename T>
hipsolverStatus_t Orgqr(
        hipsolverHandle_t /*handle*/,
        int /*m*/,
        int /*n*/,
        int /*k*/,
        T* /*a*/,
        int /*lda*/,
        T* /*tau*/,
        T* /*work*/,
        int /*lwork*/,
        int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by orgqr (QR)"};
}

template <typename T>
hipsolverStatus_t PotrfBuffersize(
        hipsolverHandle_t /*handle*/, hipblasFillMode_t /*uplo*/, int /*n*/, T* /*a*/, int /*lda*/, int* /*lwork*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by potrf (Cholesky)"};
}

template <typename T>
hipsolverStatus_t Potrf(
        hipsolverHandle_t /*handle*/,
        hipblasFillMode_t /*uplo*/,
        int /*n*/,
        T* /*a*/,
        int /*lda*/,
        T* /*workspace*/,
        int /*lwork*/,
        int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by potrf (Cholesky)"};
}

template <typename T>
hipsolverStatus_t SyevdBuffersize(
        hipsolverHandle_t /*handle*/,
        hipsolverEigMode_t /*jobz*/,
        hipblasFillMode_t /*uplo*/,
        int /*n*/,
        T* /*a*/,
        int /*lda*/,
        T* /*w*/,
        int* /*lwork*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by syevd (Eigen)"};
}

template <typename T>
hipsolverStatus_t Syevd(
        hipsolverHandle_t /*handle*/,
        hipsolverEigMode_t /*jobz*/,
        hipblasFillMode_t /*uplo*/,
        int /*n*/,
        T* /*a*/,
        int /*lda*/,
        T* /*w*/,
        T* /*work*/,
        int /*lwork*/,
        int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by syevd (Eigen)"};
}

template <>
hipsolverStatus_t GetrfBuffersize<double>(hipsolverHandle_t handle, int m, int n, double* a, int lda, int* lwork) {
    return hipsolverDnDgetrf_bufferSize(handle, m, n, a, lda, lwork);
}

template <>
hipsolverStatus_t GetrfBuffersize<float>(hipsolverHandle_t handle, int m, int n, float* a, int lda, int* lwork) {
    return hipsolverDnSgetrf_bufferSize(handle, m, n, a, lda, lwork);
}

template <>
hipsolverStatus_t Getrf<double>(hipsolverHandle_t handle, int m, int n, double* a, int lda, double* workspace, int* devipiv, int* devinfo) {
    return hipsolverDnDgetrf(handle, m, n, a, lda, workspace, devipiv, devinfo);
}

template <>
hipsolverStatus_t Getrf<float>(hipsolverHandle_t handle, int m, int n, float* a, int lda, float* workspace, int* devipiv, int* devinfo) {
    return hipsolverDnSgetrf(handle, m, n, a, lda, workspace, devipiv, devinfo);
}

template <>
hipsolverStatus_t Getrs<double>(
        hipsolverHandle_t handle,
        hipblasOperation_t trans,
        int n,
        int nrhs,
        double* a,
        int lda,
        int* devipiv,
        double* b,
        int ldb,
        int* devinfo) {
    return hipsolverDnDgetrs(handle, trans, n, nrhs, a, lda, devipiv, b, ldb, devinfo);
}

template <>
hipsolverStatus_t Getrs<float>(
        hipsolverHandle_t handle,
        hipblasOperation_t trans,
        int n,
        int nrhs,
        float* a,
        int lda,
        int* devipiv,
        float* b,
        int ldb,
        int* devinfo) {
    return hipsolverDnSgetrs(handle, trans, n, nrhs, a, lda, devipiv, b, ldb, devinfo);
}

template <>
hipsolverStatus_t GesvdBuffersize<double>(hipsolverHandle_t handle, int m, int n, int* lwork) {
    return hipsolverDnDgesvd_bufferSize(handle, m, n, lwork);
}

template <>
hipsolverStatus_t GesvdBuffersize<float>(hipsolverHandle_t handle, int m, int n, int* lwork) {
    return hipsolverDnSgesvd_bufferSize(handle, m, n, lwork);
}

template <>
hipsolverStatus_t Gesvd<double>(
        hipsolverHandle_t handle,
        signed char jobu,
        signed char jobvt,
        int m,
        int n,
        double* a,
        int lda,
        double* s,
        double* u,
        int ldu,
        double* vt,
        int ldvt,
        double* work,
        int lwork,
        double* rwork,
        int* devinfo) {
    return hipsolverDnDgesvd(handle, jobu, jobvt, m, n, a, lda, s, u, ldu, vt, ldvt, work, lwork, rwork, devinfo);
}

template <>
hipsolverStatus_t Gesvd<float>(
        hipsolverHandle_t handle,
        signed char jobu,
        signed char jobvt,
        int m,
        int n,
        float* a,
        int lda,
        float* s,
        float* u,
        int ldu,
        float* vt,
        int ldvt,
        float* work,
        int lwork,
        float* rwork,
        int* devinfo) {
    return hipsolverDnSgesvd(handle, jobu, jobvt, m, n, a, lda, s, u, ldu, vt, ldvt, work, lwork, rwork, devinfo);
}

template <>
hipsolverStatus_t GeqrfBufferSize<double>(hipsolverHandle_t handle, int m, int n, double* a, int lda, int* lwork) {
    return hipsolverDnDgeqrf_bufferSize(handle, m, n, a, lda, lwork);
}

template <>
hipsolverStatus_t GeqrfBufferSize<float>(hipsolverHandle_t handle, int m, int n, float* a, int lda, int* lwork) {
    return hipsolverDnSgeqrf_bufferSize(handle, m, n, a, lda, lwork);
}

template <>
hipsolverStatus_t Geqrf<double>(
        hipsolverHandle_t handle, int m, int n, double* a, int lda, double* tau, double* workspace, int lwork, int* devinfo) {
    return hipsolverDnDgeqrf(handle, m, n, a, lda, tau, workspace, lwork, devinfo);
}

template <>
hipsolverStatus_t Geqrf<float>(
        hipsolverHandle_t handle, int m, int n, float* a, int lda, float* tau, float* workspace, int lwork, int* devinfo) {
    return hipsolverDnSgeqrf(handle, m, n, a, lda, tau, workspace, lwork, devinfo);
}

template <>
hipsolverStatus_t OrgqrBufferSize<double>(hipsolverHandle_t handle, int m, int n, int k, double* a, int lda, double* tau, int* lwork) {
    return hipsolverDnDorgqr_bufferSize(handle, m, n, k, a, lda, tau, lwork);
}

template <>
hipsolverStatus_t OrgqrBufferSize<float>(hipsolverHandle_t handle, int m, int n, int k, float* a, int lda, float* tau, int* lwork) {
    return hipsolverDnSorgqr_bufferSize(handle, m, n, k, a, lda, tau, lwork);
}

template <>
hipsolverStatus_t Orgqr<double>(
        hipsolverHandle_t handle, int m, int n, int k, double* a, int lda, double* tau, double* work, int lwork, int* devinfo) {
    return hipsolverDnDorgqr(handle, m, n, k, a, lda, tau, work, lwork, devinfo);
}

template <>
hipsolverStatus_t Orgqr<float>(
        hipsolverHandle_t handle, int m, int n, int k, float* a, int lda, float* tau, float* work, int lwork, int* devinfo) {
    return hipsolverDnSorgqr(handle, m, n, k, a, lda, tau, work, lwork, devinfo);
}

template <>
hipsolverStatus_t PotrfBuffersize<double>(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* a, int lda, int* lwork) {
    return hipsolverDnDpotrf_bufferSize(handle, uplo, n, a, lda, lwork);
}

template <>
hipsolverStatus_t PotrfBuffersize<float>(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* a, int lda, int* lwork) {
    return hipsolverDnSpotrf_bufferSize(handle, uplo, n, a, lda, lwork);
}

template <>
hipsolverStatus_t Potrf<double>(
        hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double* a, int lda, double* workspace, int lwork, int* devinfo) {
    return hipsolverDnDpotrf(handle, uplo, n, a, lda, workspace, lwork, devinfo);
}

template <>
hipsolverStatus_t Potrf<float>(
        hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float* a, int lda, float* workspace, int lwork, int* devinfo) {
    return hipsolverDnSpotrf(handle, uplo, n, a, lda, workspace, lwork, devinfo);
}

template <>
hipsolverStatus_t SyevdBuffersize<double>(
        hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, double* a, int lda, double* w, int* lwork) {
    return hipsolverDnDsyevd_bufferSize(handle, jobz, uplo, n, a, lda, w, lwork);
}

template <>
hipsolverStatus_t SyevdBuffersize<float>(
        hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, float* a, int lda, float* w, int* lwork) {
    return hipsolverDnSsyevd_bufferSize(handle, jobz, uplo, n, a, lda, w, lwork);
}

template <>
hipsolverStatus_t Syevd<double>(
        hipsolverHandle_t handle,
        hipsolverEigMode_t jobz,
        hipblasFillMode_t uplo,
        int n,
        double* a,
        int lda,
        double* w,
        double* work,
        int lwork,
        int* devinfo) {
    return hipsolverDnDsyevd(handle, jobz, uplo, n, a, lda, w, work, lwork, devinfo);
}

template <>
hipsolverStatus_t Syevd<float>(
        hipsolverHandle_t handle,
        hipsolverEigMode_t jobz,
        hipblasFillMode_t uplo,
        int n,
        float* a,
        int lda,
        float* w,
        float* work,
        int lwork,
        int* devinfo) {
    return hipsolverDnSsyevd(handle, jobz, uplo, n, a, lda, w, work, lwork, devinfo);
}

template <typename T>
void SolveImpl(const Array& a, const Array& b, const Array& out) {
    Device& device = a.device();
    Dtype dtype = a.dtype();

    cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(static_cast<CudaDevice&>(device));

    Array lu_matrix = Empty(a.shape(), dtype, device);
    device.backend().CallKernel<CopyKernel>(a.Transpose(), lu_matrix);
    auto lu_ptr = static_cast<T*>(internal::GetRawOffsetData(lu_matrix));

    int64_t m = a.shape()[0];
    int64_t lda = std::max(int64_t{1}, m);
    int64_t nrhs = 1;
    if (b.ndim() == 2) {
        nrhs = b.shape()[1];
    }

    Array ipiv = Empty(Shape{m}, Dtype::kInt32, device);
    auto ipiv_ptr = static_cast<int*>(internal::GetRawOffsetData(ipiv));

    int buffersize = 0;
    device_internals.cusolverdn_handle().Call(GetrfBuffersize<T>, m, m, lu_ptr, lda, &buffersize);

    Array work = Empty(Shape{buffersize}, dtype, device);
    auto work_ptr = static_cast<T*>(internal::GetRawOffsetData(work));

    std::shared_ptr<void> devinfo = device.Allocate(sizeof(int));

    device_internals.cusolverdn_handle().Call(Getrf<T>, m, m, lu_ptr, lda, work_ptr, ipiv_ptr, static_cast<int*>(devinfo.get()));

    int devinfo_h = 0;
    Device& native_device = GetDefaultContext().GetDevice({"native", 0});
    device.MemoryCopyTo(&devinfo_h, devinfo.get(), sizeof(int), native_device);
    if (devinfo_h != 0) {
        throw ChainerxError{"Unsuccessful getrf (LU) execution. Info = ", devinfo_h};
    }

    Array out_transposed = b.Transpose().Copy();
    auto out_ptr = static_cast<T*>(internal::GetRawOffsetData(out_transposed));

    device_internals.cusolverdn_handle().Call(
            Getrs<T>, HIPBLAS_OP_N, m, nrhs, lu_ptr, lda, ipiv_ptr, out_ptr, lda, static_cast<int*>(devinfo.get()));

    device.MemoryCopyTo(&devinfo_h, devinfo.get(), sizeof(int), native_device);
    if (devinfo_h != 0) {
        throw ChainerxError{"Unsuccessful getrs (Solve) execution. Info = ", devinfo_h};
    }

    device.backend().CallKernel<CopyKernel>(out_transposed.Transpose(), out);
}

template <typename T>
void QrImpl(const Array& a, const Array& q, const Array& r, const Array& tau, QrMode mode) {
    Device& device = a.device();
    Dtype dtype = a.dtype();

    int64_t m = a.shape()[0];
    int64_t n = a.shape()[1];
    int64_t k = std::min(m, n);
    int64_t lda = std::max(int64_t{1}, m);

    // cuSOLVER does not return correct result in this case and older versions of cuSOLVER (<10.1)
    // might not work well with zero-sized arrays therefore it's better to return earlier
    if (a.shape().GetTotalSize() == 0) {
        if (mode == QrMode::kComplete) {
            device.backend().CallKernel<IdentityKernel>(q);
        }
        return;
    }

    Array r_temp = a.Transpose().Copy();  // QR decomposition is done in-place

    cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(static_cast<CudaDevice&>(device));

    auto r_ptr = static_cast<T*>(internal::GetRawOffsetData(r_temp));
    auto tau_ptr = static_cast<T*>(internal::GetRawOffsetData(tau));

    std::shared_ptr<void> devinfo = device.Allocate(sizeof(int));

    int buffersize_geqrf = 0;
    device_internals.cusolverdn_handle().Call(GeqrfBufferSize<T>, m, n, r_ptr, lda, &buffersize_geqrf);

    Array work = Empty(Shape{buffersize_geqrf}, dtype, device);
    auto work_ptr = static_cast<T*>(internal::GetRawOffsetData(work));

    device_internals.cusolverdn_handle().Call(
            Geqrf<T>, m, n, r_ptr, lda, tau_ptr, work_ptr, buffersize_geqrf, static_cast<int*>(devinfo.get()));

    int devinfo_h = 0;
    Device& native_device = GetDefaultContext().GetDevice({"native", 0});
    device.MemoryCopyTo(&devinfo_h, devinfo.get(), sizeof(int), native_device);
    if (devinfo_h != 0) {
        throw ChainerxError{"Unsuccessful geqrf (QR) execution. Info = ", devinfo_h};
    }

    if (mode == QrMode::kR) {
        r_temp = r_temp.At(std::vector<ArrayIndex>{Slice{}, Slice{0, k}}).Transpose();  // R = R[:, 0:k].T
        r_temp = Triu(r_temp, 0);
        device.backend().CallKernel<CopyKernel>(r_temp, r);
        return;
    }

    if (mode == QrMode::kRaw) {
        device.backend().CallKernel<CopyKernel>(r_temp, r);
        return;
    }

    int64_t mc;
    Shape q_shape{0};
    if (mode == QrMode::kComplete && m > n) {
        mc = m;
        q_shape = Shape{m, m};
    } else {
        mc = k;
        q_shape = Shape{n, m};
    }
    Array q_temp = Empty(q_shape, dtype, device);

    device.backend().CallKernel<CopyKernel>(r_temp, q_temp.At(std::vector<ArrayIndex>{Slice{0, n}, Slice{}}));  // Q[0:n, :] = R
    auto q_ptr = static_cast<T*>(internal::GetRawOffsetData(q_temp));

    int buffersize_orgqr = 0;
    device_internals.cusolverdn_handle().Call(OrgqrBufferSize<T>, m, mc, k, q_ptr, lda, tau_ptr, &buffersize_orgqr);

    Array work_orgqr = Empty(Shape{buffersize_orgqr}, dtype, device);
    auto work_orgqr_ptr = static_cast<T*>(internal::GetRawOffsetData(work_orgqr));

    device_internals.cusolverdn_handle().Call(
            Orgqr<T>, m, mc, k, q_ptr, lda, tau_ptr, work_orgqr_ptr, buffersize_orgqr, static_cast<int*>(devinfo.get()));

    device.MemoryCopyTo(&devinfo_h, devinfo.get(), sizeof(int), native_device);
    if (devinfo_h != 0) {
        throw ChainerxError{"Unsuccessful orgqr (QR) execution. Info = ", devinfo_h};
    }

    q_temp = q_temp.At(std::vector<ArrayIndex>{Slice{0, mc}, Slice{}}).Transpose();  // Q = Q[0:mc, :].T
    r_temp = r_temp.At(std::vector<ArrayIndex>{Slice{}, Slice{0, mc}}).Transpose();  // R = R[:, 0:mc].T
    r_temp = Triu(r_temp, 0);

    device.backend().CallKernel<CopyKernel>(q_temp, q);
    device.backend().CallKernel<CopyKernel>(r_temp, r);
}

}  // namespace

class CudaSolveKernel : public SolveKernel {
public:
    void Call(const Array& a, const Array& b, const Array& out) override {
        Device& device = a.device();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);
        CHAINERX_ASSERT(a.shape()[0] == a.shape()[1]);

        VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            SolveImpl<T>(a.dtype() == out.dtype() ? a : a.AsType(out.dtype()), b.dtype() == out.dtype() ? b : b.AsType(out.dtype()), out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(SolveKernel, CudaSolveKernel);

class CudaInverseKernel : public InverseKernel {
public:
    void Call(const Array& a, const Array& out) override {
        Device& device = a.device();
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);
        CHAINERX_ASSERT(a.shape()[0] == a.shape()[1]);

        // There is LAPACK routine ``getri`` for computing the inverse of an LU-factored matrix,
        // but cuSOLVER does not have it implemented, therefore inverse is obtained with ``getrs``
        // inv(A) == solve(A, Identity)
        Array b = Identity(a.shape()[0], dtype, device);
        device.backend().CallKernel<SolveKernel>(a, b, out);
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(InverseKernel, CudaInverseKernel);

class CudaSvdKernel : public SvdKernel {
public:
    void Call(const Array& a, const Array& u, const Array& s, const Array& vt, bool full_matrices, bool compute_uv) override {
        Device& device = a.device();
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);

        if (a.shape().GetTotalSize() == 0) {
            if (full_matrices && compute_uv) {
                device.backend().CallKernel<IdentityKernel>(u);
                device.backend().CallKernel<IdentityKernel>(vt);
            }
            // This kernel works correctly for zero-sized input also without early return
            return;
        }

        // cuSOLVER assumes arrays are in column-major order.
        // In order to avoid transposing the input matrix, matrix dimensions are swapped.
        // Since the input is assumed to be transposed, it is necessary to
        // swap the pointers to u and vt matrices when calling Gesvd.
        int64_t n = a.shape()[0];
        int64_t m = a.shape()[1];
        int64_t k = std::min(m, n);

        Array x = EmptyLike(a, device);
        Array u_temp{};
        Array vt_temp{};
        bool trans_flag;

        // Remark: gesvd only supports m>=n.
        // See: https://docs.nvidia.com/cuda/cusolver/index.html#cuds-lt-t-gt-gesvd
        // Therefore for the case m<n we calculuate svd of transposed matrix,
        // instead of calculating svd(A) = U S V^T, we compute svd(A^T) = V S U^T
        if (m >= n) {
            device.backend().CallKernel<CopyKernel>(a, x);
            trans_flag = false;
        } else {
            m = a.shape()[0];
            n = a.shape()[1];
            x = x.Reshape(Shape{n, m});
            device.backend().CallKernel<CopyKernel>(a.Transpose(), x);
            trans_flag = true;

            // Temporary arrays for u, vt are needed to store transposed results
            Shape u_shape;
            Shape vt_shape;
            if (compute_uv) {
                if (full_matrices) {
                    u_shape = Shape{m, m};
                    vt_shape = Shape{n, n};
                } else {
                    u_shape = Shape{k, m};
                    vt_shape = Shape{n, k};
                }
            } else {
                u_shape = Shape{0};
                vt_shape = Shape{0};
            }
            u_temp = Empty(u_shape, dtype, device);
            vt_temp = Empty(vt_shape, dtype, device);
        }

        int64_t lda = std::max(int64_t{1}, m);
        int64_t ldu = std::max(int64_t{1}, m);
        int64_t ldvt = full_matrices ? std::max(int64_t{1}, n) : std::max(int64_t{1}, k);

        auto svd_impl = [&](auto pt) {
            using T = typename decltype(pt)::type;
            cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(static_cast<CudaDevice&>(device));

            auto x_ptr = static_cast<T*>(internal::GetRawOffsetData(x));
            auto s_ptr = static_cast<T*>(internal::GetRawOffsetData(s));
            auto u_ptr = static_cast<T*>(internal::GetRawOffsetData(u));
            auto vt_ptr = static_cast<T*>(internal::GetRawOffsetData(vt));
            if (trans_flag) {
                u_ptr = static_cast<T*>(internal::GetRawOffsetData(vt_temp));
                vt_ptr = static_cast<T*>(internal::GetRawOffsetData(u_temp));
            }

            std::shared_ptr<void> devinfo = device.Allocate(sizeof(int));

            int buffersize = 0;
            device_internals.cusolverdn_handle().Call(GesvdBuffersize<T>, m, n, &buffersize);

            Array work = Empty(Shape{buffersize}, dtype, device);
            auto work_ptr = static_cast<T*>(internal::GetRawOffsetData(work));

            signed char job;
            if (compute_uv) {
                job = full_matrices ? 'A' : 'S';
            } else {
                job = 'N';
            }

            // When calling Gesvd pointers to u and vt are swapped instead of transposing the input matrix.
            device_internals.cusolverdn_handle().Call(
                    Gesvd<T>,
                    job,
                    job,
                    m,
                    n,
                    x_ptr,
                    lda,
                    s_ptr,
                    vt_ptr,
                    ldu,
                    u_ptr,
                    ldvt,
                    work_ptr,
                    buffersize,
                    nullptr,
                    static_cast<int*>(devinfo.get()));

            int devinfo_h = 0;
            Device& native_device = GetDefaultContext().GetDevice({"native", 0});
            device.MemoryCopyTo(&devinfo_h, devinfo.get(), sizeof(int), native_device);
            if (devinfo_h != 0) {
                throw ChainerxError{"Unsuccessful gesvd (SVD) execution. Info = ", devinfo_h};
            }

            if (trans_flag) {
                device.backend().CallKernel<CopyKernel>(u_temp.Transpose(), u);
                device.backend().CallKernel<CopyKernel>(vt_temp.Transpose(), vt);
            }
        };

        VisitFloatingPointDtype(dtype, svd_impl);
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(SvdKernel, CudaSvdKernel);

class CudaQrKernel : public QrKernel {
public:
    void Call(const Array& a, const Array& q, const Array& r, const Array& tau, QrMode mode) override {
        Device& device = a.device();
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);

        VisitFloatingPointDtype(dtype, [&](auto pt) {
            using T = typename decltype(pt)::type;
            QrImpl<T>(a, q, r, tau, mode);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(QrKernel, CudaQrKernel);

class CudaCholeskyKernel : public CholeskyKernel {
public:
    void Call(const Array& a, const Array& out) override {
        Device& device = a.device();
        device.CheckDevicesCompatible(a, out);
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);
        CHAINERX_ASSERT(out.ndim() == 2);
        CHAINERX_ASSERT(a.shape()[0] == a.shape()[1]);
        CHAINERX_ASSERT(out.IsContiguous());
        CHAINERX_ASSERT(a.dtype() == out.dtype());

        // cuSOLVER might not work well with zero-sized arrays for older versions of cuSOLVER (<10.1)
        // therefore it's better to return earlier
        if (a.shape().GetTotalSize() == 0) {
            return;
        }

        // potrf (cholesky) stores result in-place, therefore copy ``a`` to ``out`` and then pass ``out`` to the routine
        device.backend().CallKernel<CopyKernel>(Tril(a, 0), out);

        auto cholesky_impl = [&](auto pt) {
            using T = typename decltype(pt)::type;

            // Note that cuSOLVER uses Fortran order.
            // To compute a lower triangular matrix L = cholesky(A), we use cuSOLVER to compute an upper triangular matrix U = cholesky(A).
            hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;

            cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(static_cast<CudaDevice&>(device));

            // compute workspace size and prepare workspace
            auto out_ptr = static_cast<T*>(internal::GetRawOffsetData(out));
            int work_size = 0;
            int64_t n = a.shape()[0];
            device_internals.cusolverdn_handle().Call(PotrfBuffersize<T>, uplo, n, out_ptr, std::max(int64_t{1}, n), &work_size);

            // POTRF execution
            Array work = Empty(Shape{work_size}, dtype, device);
            auto work_ptr = static_cast<T*>(internal::GetRawOffsetData(work));

            std::shared_ptr<void> devinfo = device.Allocate(sizeof(int));
            device_internals.cusolverdn_handle().Call(
                    Potrf<T>, uplo, n, out_ptr, std::max(int64_t{1}, n), work_ptr, work_size, static_cast<int*>(devinfo.get()));

            int devinfo_h = 0;
            Device& native_device = GetDefaultContext().GetDevice({"native", 0});
            device.MemoryCopyTo(&devinfo_h, devinfo.get(), sizeof(int), native_device);
            if (devinfo_h != 0) {
                throw ChainerxError{"Unsuccessful potrf (Cholesky) execution. Info = ", devinfo_h};
            }
        };

        VisitFloatingPointDtype(dtype, cholesky_impl);
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(CholeskyKernel, CudaCholeskyKernel);

class CudaSyevdKernel : public SyevdKernel {
public:
    void Call(const Array& a, const Array& w, const Array& v, char uplo, bool compute_v) override {
        Device& device = a.device();
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);

        device.backend().CallKernel<CopyKernel>(a, v);

        int64_t m = a.shape()[0];
        int64_t n = a.shape()[1];

        auto syevd_impl = [&](auto pt) {
            using T = typename decltype(pt)::type;
            cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(static_cast<CudaDevice&>(device));

            auto v_ptr = static_cast<T*>(internal::GetRawOffsetData(v));
            auto w_ptr = static_cast<T*>(internal::GetRawOffsetData(w));

            hipsolverEigMode_t jobz = compute_v ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;

            // cuSOLVER assumes that arrays are stored in column-major order
            // The uplo argument is swapped instead of transposing the input matrix
            hipblasFillMode_t uplo_cublas = toupper(uplo) == 'U' ? HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER;

            int buffersize = 0;
            // When calling Syevd matrix dimensions are swapped instead of transposing the input matrix
            device_internals.cusolverdn_handle().Call(
                    SyevdBuffersize<T>, jobz, uplo_cublas, n, v_ptr, std::max(int64_t{1}, m), w_ptr, &buffersize);

            Array work = Empty(Shape{buffersize}, dtype, device);
            auto work_ptr = static_cast<T*>(internal::GetRawOffsetData(work));

            std::shared_ptr<void> devinfo = device.Allocate(sizeof(int));

            device_internals.cusolverdn_handle().Call(
                    Syevd<T>,
                    jobz,
                    uplo_cublas,
                    n,
                    v_ptr,
                    std::max(int64_t{1}, m),
                    w_ptr,
                    work_ptr,
                    buffersize,
                    static_cast<int*>(devinfo.get()));

            int devinfo_h = 0;
            Device& native_device = GetDefaultContext().GetDevice({"native", 0});
            device.MemoryCopyTo(&devinfo_h, devinfo.get(), sizeof(int), native_device);
            if (devinfo_h != 0) {
                throw ChainerxError{"Unsuccessful syevd (Eigen Decomposition) execution. Info = ", devinfo_h};
            }

            // v is stored now in column-major order, need to transform it to row-major
            device.backend().CallKernel<CopyKernel>(v.Transpose(), v);
        };

        VisitFloatingPointDtype(dtype, syevd_impl);
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(SyevdKernel, CudaSyevdKernel);

}  // namespace cuda
}  // namespace chainerx
