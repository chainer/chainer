#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <cstdint>
#include <mutex>
#include <type_traits>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <cuda_fp16.hpp>

#include "chainerx/array.h"
#include "chainerx/axes.h"
#include "chainerx/backend.h"
#include "chainerx/backend_util.h"
#include "chainerx/cuda/hipblas.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/cusolver.h"
#include "chainerx/cuda/data_type.cuh"
#include "chainerx/cuda/float16.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/error.h"
#include "chainerx/float16.h"
#include "chainerx/kernels/creation.h"
#include "chainerx/kernels/linalg.h"
#include "chainerx/kernels/misc.h"
#include "chainerx/macro.h"
#include "chainerx/native/native_device.h"
#include "chainerx/routines/arithmetic.h"
#include "chainerx/routines/creation.h"
#include "chainerx/routines/indexing.h"
#include "chainerx/routines/linalg.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
hipsolverStatus_t GetrfBuffersize(hipsolverHandle_t /*handle*/, int /*m*/, int /*n*/, T* /*a*/, int /*lda*/, int* /*lwork*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by getrf (LU)"};
}

template <typename T>
hipsolverStatus_t Getrf(
        hipsolverHandle_t /*handle*/, int /*m*/, int /*n*/, T* /*a*/, int /*lda*/, T* /*workspace*/, int* /*devipiv*/, int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by getrf (LU)"};
}

template <typename T>
hipsolverStatus_t Getrs(
        hipsolverHandle_t /*handle*/,
        hipblasOperation_t /*trans*/,
        int /*n*/,
        int /*nrhs*/,
        T* /*a*/,
        int /*lda*/,
        int* /*devipiv*/,
        T* /*b*/,
        int /*ldb*/,
        int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by getrs (Solve)"};
}

template <typename T>
hipsolverStatus_t GesvdBuffersize(hipsolverHandle_t /*handle*/, int /*m*/, int /*n*/, int* /*lwork*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by gesvd (SVD)"};
}

template <typename T>
hipsolverStatus_t Gesvd(
        hipsolverHandle_t /*handle*/,
        signed char /*jobu*/,
        signed char /*jobvt*/,
        int /*m*/,
        int /*n*/,
        T* /*a*/,
        int /*lda*/,
        T* /*s*/,
        T* /*u*/,
        int /*ldu*/,
        T* /*vt*/,
        int /*ldvt*/,
        T* /*work*/,
        int /*lwork*/,
        T* /*rwork*/,
        int* /*devinfo*/) {
    throw DtypeError{"Only Arrays of float or double type are supported by gesvd (SVD)"};
}

template <>
hipsolverStatus_t GetrfBuffersize<double>(hipsolverHandle_t handle, int m, int n, double* a, int lda, int* lwork) {
    return hipsolverDnDgetrf_bufferSize(handle, m, n, a, lda, lwork);
}

template <>
hipsolverStatus_t GetrfBuffersize<float>(hipsolverHandle_t handle, int m, int n, float* a, int lda, int* lwork) {
    return hipsolverDnSgetrf_bufferSize(handle, m, n, a, lda, lwork);
}

template <>
hipsolverStatus_t Getrf<double>(hipsolverHandle_t handle, int m, int n, double* a, int lda, double* workspace, int* devipiv, int* devinfo) {
    return hipsolverDnDgetrf(handle, m, n, a, lda, workspace, devipiv, devinfo);
}

template <>
hipsolverStatus_t Getrf<float>(hipsolverHandle_t handle, int m, int n, float* a, int lda, float* workspace, int* devipiv, int* devinfo) {
    return hipsolverDnSgetrf(handle, m, n, a, lda, workspace, devipiv, devinfo);
}

template <>
hipsolverStatus_t Getrs<double>(
        hipsolverHandle_t handle,
        hipblasOperation_t trans,
        int n,
        int nrhs,
        double* a,
        int lda,
        int* devipiv,
        double* b,
        int ldb,
        int* devinfo) {
    return hipsolverDnDgetrs(handle, trans, n, nrhs, a, lda, devipiv, b, ldb, devinfo);
}

template <>
hipsolverStatus_t Getrs<float>(
        hipsolverHandle_t handle,
        hipblasOperation_t trans,
        int n,
        int nrhs,
        float* a,
        int lda,
        int* devipiv,
        float* b,
        int ldb,
        int* devinfo) {
    return hipsolverDnSgetrs(handle, trans, n, nrhs, a, lda, devipiv, b, ldb, devinfo);
}

template <>
hipsolverStatus_t GesvdBuffersize<double>(hipsolverHandle_t handle, int m, int n, int* lwork) {
    return hipsolverDnDgesvd_bufferSize(handle, m, n, lwork);
}

template <>
hipsolverStatus_t GesvdBuffersize<float>(hipsolverHandle_t handle, int m, int n, int* lwork) {
    return hipsolverDnSgesvd_bufferSize(handle, m, n, lwork);
}

template <>
hipsolverStatus_t Gesvd<double>(
        hipsolverHandle_t handle,
        signed char jobu,
        signed char jobvt,
        int m,
        int n,
        double* a,
        int lda,
        double* s,
        double* u,
        int ldu,
        double* vt,
        int ldvt,
        double* work,
        int lwork,
        double* rwork,
        int* devinfo) {
    return hipsolverDnDgesvd(handle, jobu, jobvt, m, n, a, lda, s, u, ldu, vt, ldvt, work, lwork, rwork, devinfo);
}

template <>
hipsolverStatus_t Gesvd<float>(
        hipsolverHandle_t handle,
        signed char jobu,
        signed char jobvt,
        int m,
        int n,
        float* a,
        int lda,
        float* s,
        float* u,
        int ldu,
        float* vt,
        int ldvt,
        float* work,
        int lwork,
        float* rwork,
        int* devinfo) {
    return hipsolverDnSgesvd(handle, jobu, jobvt, m, n, a, lda, s, u, ldu, vt, ldvt, work, lwork, rwork, devinfo);
}

template <typename T>
void SolveImpl(const Array& a, const Array& b, const Array& out) {
    Device& device = a.device();
    Dtype dtype = a.dtype();

    cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(static_cast<CudaDevice&>(device));

    Array lu_matrix = Empty(a.shape(), dtype, device);
    device.backend().CallKernel<CopyKernel>(a.Transpose(), lu_matrix);
    auto lu_ptr = static_cast<T*>(internal::GetRawOffsetData(lu_matrix));

    int64_t m = a.shape()[0];
    int64_t nrhs = 1;
    if (b.ndim() == 2) {
        nrhs = b.shape()[1];
    }

    Array ipiv = Empty(Shape{m}, Dtype::kInt32, device);
    auto ipiv_ptr = static_cast<int*>(internal::GetRawOffsetData(ipiv));

    int buffersize = 0;
    device_internals.cusolverdn_handle().Call(GetrfBuffersize<T>, m, m, lu_ptr, m, &buffersize);

    Array work = Empty(Shape{buffersize}, dtype, device);
    auto work_ptr = static_cast<T*>(internal::GetRawOffsetData(work));

    std::shared_ptr<void> devinfo = device.Allocate(sizeof(int));

    device_internals.cusolverdn_handle().Call(Getrf<T>, m, m, lu_ptr, m, work_ptr, ipiv_ptr, static_cast<int*>(devinfo.get()));

    int devinfo_h = 0;
    Device& native_device = GetDefaultContext().GetDevice({"native", 0});
    device.MemoryCopyTo(&devinfo_h, devinfo.get(), sizeof(int), native_device);
    if (devinfo_h != 0) {
        throw ChainerxError{"Unsuccessful getrf (LU) execution. Info = ", devinfo_h};
    }

    Array out_transposed = b.Transpose().Copy();
    auto out_ptr = static_cast<T*>(internal::GetRawOffsetData(out_transposed));

    device_internals.cusolverdn_handle().Call(
            Getrs<T>, HIPBLAS_OP_N, m, nrhs, lu_ptr, m, ipiv_ptr, out_ptr, m, static_cast<int*>(devinfo.get()));

    device.MemoryCopyTo(&devinfo_h, devinfo.get(), sizeof(int), native_device);
    if (devinfo_h != 0) {
        throw ChainerxError{"Unsuccessful getrs (Solve) execution. Info = ", devinfo_h};
    }

    device.backend().CallKernel<CopyKernel>(out_transposed.Transpose(), out);
}

}  // namespace

class CudaSolveKernel : public SolveKernel {
public:
    void Call(const Array& a, const Array& b, const Array& out) override {
        Device& device = a.device();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);
        CHAINERX_ASSERT(a.shape()[0] == a.shape()[1]);

        VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            SolveImpl<T>(a.dtype() == out.dtype() ? a : a.AsType(out.dtype()), b.dtype() == out.dtype() ? b : b.AsType(out.dtype()), out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(SolveKernel, CudaSolveKernel);

class CudaInverseKernel : public InverseKernel {
public:
    void Call(const Array& a, const Array& out) override {
        Device& device = a.device();
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);
        CHAINERX_ASSERT(a.shape()[0] == a.shape()[1]);

        // There is LAPACK routine ``getri`` for computing the inverse of an LU-factored matrix,
        // but cuSOLVER does not have it implemented, therefore inverse is obtained with ``getrs``
        // inv(A) == solve(A, Identity)
        Array b = Identity(a.shape()[0], dtype, device);
        device.backend().CallKernel<SolveKernel>(a, b, out);
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(InverseKernel, CudaInverseKernel);

class CudaSvdKernel : public SvdKernel {
public:
    void Call(const Array& a, const Array& u, const Array& s, const Array& vt, bool full_matrices) override {
        Device& device = a.device();
        Dtype dtype = a.dtype();
        CudaSetDeviceScope scope{device.index()};

        CHAINERX_ASSERT(a.ndim() == 2);

        bool compute_uv = u.shape()[0] != 0 && vt.shape()[0] != 0;

        // cuSOLVER assumes arrays are in column-major order.
        // In order to avoid transposing the input matrix, matrix dimensions are swapped.
        // Since the input is assumed to be transposed, it is necessary to
        // swap the pointers to u and vt matrices when calling Gesvd.
        int64_t n = a.shape()[0];
        int64_t m = a.shape()[1];
        int64_t k = std::min(m, n);

        Array x = EmptyLike(a, device);
        Array u_temp{};
        Array vt_temp{};
        bool trans_flag;

        // Remark: gesvd only supports m>=n.
        // See: https://docs.nvidia.com/cuda/cusolver/index.html#cuds-lt-t-gt-gesvd
        // Therefore for the case m<n we calculuate svd of transposed matrix,
        // instead of calculating svd(A) = U S V^T, we compute svd(A^T) = V S U^T
        if (m >= n) {
            device.backend().CallKernel<CopyKernel>(a, x);
            trans_flag = false;
        } else {
            m = a.shape()[0];
            n = a.shape()[1];
            x = x.Reshape(Shape{n, m});
            device.backend().CallKernel<CopyKernel>(a.Transpose(), x);
            trans_flag = true;

            // Temporary arrays for u, vt are needed to store transposed results
            Shape u_shape;
            Shape vt_shape;
            if (compute_uv) {
                if (full_matrices) {
                    u_shape = Shape{m, m};
                    vt_shape = Shape{n, n};
                } else {
                    u_shape = Shape{k, m};
                    vt_shape = Shape{n, k};
                }
            } else {
                u_shape = Shape{0};
                vt_shape = Shape{0};
            }
            u_temp = Empty(u_shape, dtype, device);
            vt_temp = Empty(vt_shape, dtype, device);
        }

        int64_t ldu = m;
        int64_t ldvt = full_matrices ? n : k;

        auto svd_impl = [&](auto pt) {
            using T = typename decltype(pt)::type;
            cuda_internal::DeviceInternals& device_internals = cuda_internal::GetDeviceInternals(static_cast<CudaDevice&>(device));

            auto x_ptr = static_cast<T*>(internal::GetRawOffsetData(x));
            auto s_ptr = static_cast<T*>(internal::GetRawOffsetData(s));
            auto u_ptr = static_cast<T*>(internal::GetRawOffsetData(u));
            auto vt_ptr = static_cast<T*>(internal::GetRawOffsetData(vt));
            if (trans_flag) {
                u_ptr = static_cast<T*>(internal::GetRawOffsetData(vt_temp));
                vt_ptr = static_cast<T*>(internal::GetRawOffsetData(u_temp));
            }

            std::shared_ptr<void> devInfo = device.Allocate(sizeof(int));

            int buffersize = 0;
            device_internals.cusolverdn_handle().Call(GesvdBuffersize<T>, m, n, &buffersize);

            Array work = Empty(Shape{buffersize}, dtype, device);
            auto work_ptr = static_cast<T*>(internal::GetRawOffsetData(work));

            signed char job;
            if (compute_uv) {
                job = full_matrices ? 'A' : 'S';
            } else {
                job = 'N';
            }

            // When calling Gesvd pointers to u and vt are swapped instead of transposing the input matrix.
            device_internals.cusolverdn_handle().Call(
                    Gesvd<T>,
                    job,
                    job,
                    m,
                    n,
                    x_ptr,
                    m,
                    s_ptr,
                    vt_ptr,
                    ldu,
                    u_ptr,
                    ldvt,
                    work_ptr,
                    buffersize,
                    nullptr,
                    static_cast<int*>(devInfo.get()));

            int devInfo_h = 0;
            Device& native_device = GetDefaultContext().GetDevice({"native", 0});
            device.MemoryCopyTo(&devInfo_h, devInfo.get(), sizeof(int), native_device);
            if (devInfo_h != 0) {
                throw ChainerxError{"Unsuccessful gesvd (SVD) execution. Info = ", devInfo_h};
            }

            if (trans_flag) {
                device.backend().CallKernel<CopyKernel>(u_temp.Transpose(), u);
                device.backend().CallKernel<CopyKernel>(vt_temp.Transpose(), vt);
            }
        };

        VisitFloatingPointDtype(dtype, svd_impl);
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(SvdKernel, CudaSvdKernel);

}  // namespace cuda
}  // namespace chainerx
