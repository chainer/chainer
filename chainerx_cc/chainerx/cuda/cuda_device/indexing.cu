#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <algorithm>
#include <cstdint>
#include <mutex>
#include <numeric>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/axes.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/indexable_array.h"
#include "chainerx/indexer.h"
#include "chainerx/macro.h"
#include "chainerx/shape.h"

namespace chainerx {
namespace cuda {

namespace {

// Makes axes for permutation that moves [first_axis, last_axis) to the head.
Axes MakeRollingPermutation(int8_t first_axis, int8_t last_axis, int8_t ndim) {
    CHAINERX_ASSERT(0 <= first_axis);
    CHAINERX_ASSERT(first_axis < last_axis);
    CHAINERX_ASSERT(last_axis <= ndim);

    Axes permutation{};
    permutation.resize(ndim);
    auto head_end = permutation.begin() + (last_axis - first_axis);
    auto last = permutation.begin() + last_axis;
    std::iota(permutation.begin(), head_end, first_axis);
    std::iota(head_end, last, int8_t{0});
    std::iota(last, permutation.end(), last_axis);
    return permutation;
}

template <typename T>
__global__ void TakeKernel(
        IndexableArray<const T> a_iarray,
        IndexableArray<T> out_iarray,
        IndexableArray<const int64_t> indices_iarray,
        Indexer<> a_indexer,
        Indexer<> out_indexer,
        Indexer<> indices_indexer,
        int64_t common_total_size,
        int64_t axis_dim) {
    for (auto it = out_indexer.It(blockIdx.x * blockDim.x + threadIdx.x, blockDim.x * gridDim.x); it; ++it) {
        int64_t indices_pos = it.raw_index() / common_total_size;
        int64_t common_pos = it.raw_index() % common_total_size;

        int64_t index = indices_iarray[indices_indexer.It(indices_pos)];
        if (index < 0) {
            index = axis_dim - ((-index + axis_dim - 1) % axis_dim + 1);
        } else {
            index = index % axis_dim;
        }
        CHAINERX_ASSERT(0 <= index);
        CHAINERX_ASSERT(index < axis_dim);

        out_iarray[it] = a_iarray[a_indexer.It(index * common_total_size + common_pos)];
    }
}

template <typename T>
__global__ void AddAtKernel(
        IndexableArray<const T> a_iarray,
        IndexableArray<const T> b_iarray,
        IndexableArray<T> out_iarray,
        IndexableArray<const int64_t> indices_iarray,
        Indexer<> b_indexer,
        Indexer<> out_indexer,
        Indexer<> indices_indexer,
        int64_t common_total_size,
        int64_t axis_dim) {
    for (auto it = out_indexer.It(blockIdx.x * blockDim.x + threadIdx.x, blockDim.x * gridDim.x); it; ++it) {
        int64_t axis_pos = it.raw_index() / common_total_size;
        int64_t common_pos = it.raw_index() % common_total_size;

        T out_value = a_iarray[it];

        for (auto it_indices = indices_indexer.It(0); it_indices; ++it_indices) {
            int64_t index = indices_iarray[it_indices];

            if (index < 0) {
                index = axis_dim - ((-index + axis_dim - 1) % axis_dim + 1);
            } else {
                index = index % axis_dim;
            }
            CHAINERX_ASSERT(0 <= index);
            CHAINERX_ASSERT(index < axis_dim);

            if (index == axis_pos) {
                out_value += b_iarray[b_indexer.It(it_indices.raw_index() * common_total_size + common_pos)];
            }
        }

        out_iarray[it] = out_value;
    }
}

}  // namespace

void CudaDevice::Take(const Array& a, const Array& indices, int8_t axis, const Array& out) {
    CheckDevicesCompatible(a, indices, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;

        // a and out are transposed as follows.
        // a:       (Ni..., N,     Nj...) => (N,     Ni..., Nj...)
        // out:     (Ni..., Nk..., Nj...) => (Nk..., Ni..., Nj...)
        //
        // indices is used as is.
        // indices: (Nk...)

        IndexableArray<const T> a_iarray{a};
        Axes a_perm = MakeRollingPermutation(axis, axis + 1, a.ndim());
        a_iarray.Permute(a_perm);
        Shape a_shape = internal::TransposeShape(a.shape(), a_perm);
        Indexer<> a_indexer{a_shape};

        IndexableArray<T> out_iarray{out};
        Axes out_perm = MakeRollingPermutation(axis, axis + indices.ndim(), out.ndim());
        out_iarray.Permute(out_perm);
        Shape out_shape = internal::TransposeShape(out.shape(), out_perm);
        Indexer<> out_indexer{out_shape};

        IndexableArray<const int64_t> indices_iarray{indices};
        Indexer<> indices_indexer{indices.shape()};

        // size of (Ni..., Nj...) part
        int64_t common_total_size = a_indexer.total_size() / a_shape[0];

        // TODO(niboshi): Calculate kMaxBlockSize per device
        std::lock_guard<std::mutex> lock{*cuda_internal::g_mutex};
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&TakeKernel<T>).block_size;
        int64_t total_size = out_indexer.total_size();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        TakeKernel<<<grid_size, block_size>>>(
                a_iarray, out_iarray, indices_iarray, a_indexer, out_indexer, indices_indexer, common_total_size, a_shape[0]);
    });
}

void CudaDevice::AddAt(const Array& a, const Array& indices, int8_t axis, const Array& b, const Array& out) {
    // TODO(niboshi): Current implementation only distributes output elements in respective threads. Summation on the indices is performed
    // serially in each thread. This implementation can be improved by distributing indices as well, possibly using atomicAdd.

    CHAINERX_ASSERT(a.shape() == out.shape());
    CheckDevicesCompatible(a, indices, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;

        // b and out are transposed as follows.
        // a:       (Ni..., N,     Nj...) => (N,     Ni..., Nj...)
        // b:       (Ni..., Nk..., Nj...) => (Nk..., Ni..., Nj...)
        // out:     (Ni..., N    , Nj...) => (N    , Ni..., Nj...)
        //
        // indices is used as is.
        // indices: (Nk...)

        IndexableArray<const T> a_iarray{a};
        Axes a_perm = MakeRollingPermutation(axis, axis + 1, a.ndim());
        a_iarray.Permute(a_perm);
        Shape a_shape = internal::TransposeShape(a.shape(), a_perm);
        Indexer<> a_indexer{a_shape};

        IndexableArray<const T> b_iarray{b};
        Axes b_perm = MakeRollingPermutation(axis, axis + indices.ndim(), b.ndim());
        b_iarray.Permute(b_perm);
        Shape b_shape = internal::TransposeShape(b.shape(), b_perm);
        Indexer<> b_indexer{b_shape};

        IndexableArray<T> out_iarray{out};
        Axes out_perm = MakeRollingPermutation(axis, axis + 1, out.ndim());
        out_iarray.Permute(out_perm);
        Shape out_shape = internal::TransposeShape(out.shape(), out_perm);
        Indexer<> out_indexer{out_shape};

        IndexableArray<const int64_t> indices_iarray{indices};
        Indexer<> indices_indexer{indices.shape()};

        // size of (Ni..., Nj...) part
        int64_t common_total_size = a_indexer.total_size() / a_shape[0];

        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&AddAtKernel<T>).block_size;
        int64_t total_size = out_indexer.total_size();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        AddAtKernel<<<grid_size, block_size>>>(
                a_iarray, b_iarray, out_iarray, indices_iarray, b_indexer, out_indexer, indices_indexer, common_total_size, a_shape[0]);
    });
}

}  // namespace cuda
}  // namespace chainerx
