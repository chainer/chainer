#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <algorithm>
#include <cstdint>
#include <mutex>
#include <numeric>
#include <type_traits>

#include <gsl/gsl>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/axes.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/data_type.cuh"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/indexable_array.h"
#include "chainerx/indexer.h"
#include "chainerx/kernels/indexing.h"
#include "chainerx/macro.h"
#include "chainerx/routines/indexing.h"
#include "chainerx/shape.h"

namespace chainerx {
namespace cuda {
namespace {

// Makes axes for permutation that moves [first_axis, last_axis) to the head.
Axes MakeRollingPermutation(int8_t first_axis, int8_t last_axis, int8_t ndim) {
    CHAINERX_ASSERT(0 <= first_axis);
    CHAINERX_ASSERT(first_axis < last_axis);
    CHAINERX_ASSERT(last_axis <= ndim);

    Axes permutation{};
    permutation.resize(ndim);
    auto head_end = permutation.begin() + (last_axis - first_axis);
    auto last = permutation.begin() + last_axis;
    std::iota(permutation.begin(), head_end, first_axis);
    std::iota(head_end, last, int8_t{0});
    std::iota(last, permutation.end(), last_axis);
    return permutation;
}

template <typename T, typename TIndex>
__global__ void TakeCudaKernel(
        IndexableArray<const T> a_iarray,
        IndexableArray<T> out_iarray,
        IndexableArray<const TIndex> indices_iarray,
        Indexer<> a_indexer,
        Indexer<> out_indexer,
        Indexer<> indices_indexer,
        TIndex common_total_size,
        TIndex axis_dim) {
    static_assert(std::is_same<TIndex, int64_t>::value || std::is_same<TIndex, int32_t>::value, "");
    for (auto it = out_indexer.It(blockIdx.x * blockDim.x + threadIdx.x, blockDim.x * gridDim.x); it; ++it) {
        TIndex indices_pos = static_cast<TIndex>(it.raw_index()) / common_total_size;
        TIndex common_pos = static_cast<TIndex>(it.raw_index()) % common_total_size;

        TIndex index = indices_iarray[indices_indexer.It(indices_pos)];
        if (index < 0) {
            index = axis_dim - ((-index + axis_dim - 1) % axis_dim + 1);
        } else {
            index = index % axis_dim;
        }
        CHAINERX_ASSERT(0 <= index);
        CHAINERX_ASSERT(index < axis_dim);

        out_iarray[it] = a_iarray[a_indexer.It(index * common_total_size + common_pos)];
    }
}

template <typename T, typename TIndex>
__global__ void AddAtCudaKernel(
        IndexableArray<const T> a_iarray,
        IndexableArray<const T> b_iarray,
        IndexableArray<T> out_iarray,
        IndexableArray<const TIndex> indices_iarray,
        Indexer<> b_indexer,
        Indexer<> out_indexer,
        Indexer<> indices_indexer,
        TIndex common_total_size,
        TIndex axis_dim) {
    static_assert(std::is_same<TIndex, int64_t>::value || std::is_same<TIndex, int32_t>::value, "");
    for (auto it = out_indexer.It(blockIdx.x * blockDim.x + threadIdx.x, blockDim.x * gridDim.x); it; ++it) {
        TIndex axis_pos = static_cast<TIndex>(it.raw_index()) / common_total_size;
        TIndex common_pos = static_cast<TIndex>(it.raw_index()) % common_total_size;

        cuda_internal::DataType<T> out_value = cuda_internal::StorageToDataType<const T>(a_iarray[it]);

        for (auto it_indices = indices_indexer.It(0); it_indices; ++it_indices) {
            TIndex index = indices_iarray[it_indices];

            if (index < 0) {
                index = axis_dim - ((-index + axis_dim - 1) % axis_dim + 1);
            } else {
                index = index % axis_dim;
            }
            CHAINERX_ASSERT(0 <= index);
            CHAINERX_ASSERT(index < axis_dim);

            if (index == axis_pos) {
                out_value += cuda_internal::StorageToDataType<const T>(
                        b_iarray[b_indexer.It(it_indices.raw_index() * common_total_size + common_pos)]);
            }
        }

        out_iarray[it] = cuda_internal::DataToStorageType<T>(out_value);
    }
}

template <typename TIndex>
void TakeImpl(Device& device, const Array& a, const Array& indices, int8_t axis, const Array& out) {
    static_assert(std::is_same<TIndex, int64_t>::value || std::is_same<TIndex, int32_t>::value, "");
    CHAINERX_ASSERT(
            (std::is_same<TIndex, int64_t>::value && indices.dtype() == Dtype::kInt64) ||
            (std::is_same<TIndex, int32_t>::value && indices.dtype() == Dtype::kInt32));
    device.CheckDevicesCompatible(a, indices, out);

    CudaSetDeviceScope scope{device.index()};

    VisitDtype(out.dtype(), [&a, &indices, axis, &out](auto pt) {
        using T = typename decltype(pt)::type;

        // a and out are transposed as follows.
        // a:       (Ni..., N,     Nj...) => (N,     Ni..., Nj...)
        // out:     (Ni..., Nk..., Nj...) => (Nk..., Ni..., Nj...)
        //
        // indices is used as is.
        // indices: (Nk...)

        IndexableArray<const T> a_iarray{a};
        Axes a_perm = MakeRollingPermutation(axis, axis + 1, a.ndim());
        a_iarray.Permute(a_perm);
        Shape a_shape = internal::TransposeShape(a.shape(), a_perm);
        Indexer<> a_indexer{a_shape};

        IndexableArray<T> out_iarray{out};
        Axes out_perm = MakeRollingPermutation(axis, axis + indices.ndim(), out.ndim());
        out_iarray.Permute(out_perm);
        Shape out_shape = internal::TransposeShape(out.shape(), out_perm);
        Indexer<> out_indexer{out_shape};

        IndexableArray<const TIndex> indices_iarray{indices};
        Indexer<> indices_indexer{indices.shape()};

        // size of (Ni..., Nj...) part
        TIndex common_total_size = gsl::narrow<TIndex>(a_indexer.total_size() / a_shape[0]);

        TIndex axis_dim = gsl::narrow<TIndex>(a_shape[0]);

        // TODO(niboshi): Calculate kMaxBlockSize per device
        std::lock_guard<std::mutex> lock{*cuda_internal::g_mutex};
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&TakeCudaKernel<T, TIndex>).block_size;
        int64_t total_size = out_indexer.total_size();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<TIndex>(total_size, kMaxBlockSize);

        TakeCudaKernel<<<grid_size, block_size>>>(
                a_iarray, out_iarray, indices_iarray, a_indexer, out_indexer, indices_indexer, common_total_size, axis_dim);
    });
}

template <typename TIndex>
void AddAtImpl(Device& device, const Array& a, const Array& indices, int8_t axis, const Array& b, const Array& out) {
    // TODO(niboshi): Current implementation only distributes output elements in respective threads. Summation on the indices is performed
    // serially in each thread. This implementation can be improved by distributing indices as well, possibly using atomicAdd.

    static_assert(std::is_same<TIndex, int64_t>::value || std::is_same<TIndex, int32_t>::value, "");
    CHAINERX_ASSERT(
            (std::is_same<TIndex, int64_t>::value && indices.dtype() == Dtype::kInt64) ||
            (std::is_same<TIndex, int32_t>::value && indices.dtype() == Dtype::kInt32));
    CHAINERX_ASSERT(a.shape() == out.shape());
    device.CheckDevicesCompatible(a, indices, out);

    CudaSetDeviceScope scope{device.index()};

    VisitDtype(out.dtype(), [&a, &indices, axis, &b, &out](auto pt) {
        using T = typename decltype(pt)::type;

        // b and out are transposed as follows.
        // a:       (Ni..., N,     Nj...) => (N,     Ni..., Nj...)
        // b:       (Ni..., Nk..., Nj...) => (Nk..., Ni..., Nj...)
        // out:     (Ni..., N    , Nj...) => (N    , Ni..., Nj...)
        //
        // indices is used as is.
        // indices: (Nk...)

        IndexableArray<const T> a_iarray{a};
        Axes a_perm = MakeRollingPermutation(axis, axis + 1, a.ndim());
        a_iarray.Permute(a_perm);
        Shape a_shape = internal::TransposeShape(a.shape(), a_perm);
        Indexer<> a_indexer{a_shape};

        IndexableArray<const T> b_iarray{b};
        Axes b_perm = MakeRollingPermutation(axis, axis + indices.ndim(), b.ndim());
        b_iarray.Permute(b_perm);
        Shape b_shape = internal::TransposeShape(b.shape(), b_perm);
        Indexer<> b_indexer{b_shape};

        IndexableArray<T> out_iarray{out};
        Axes out_perm = MakeRollingPermutation(axis, axis + 1, out.ndim());
        out_iarray.Permute(out_perm);
        Shape out_shape = internal::TransposeShape(out.shape(), out_perm);
        Indexer<> out_indexer{out_shape};

        IndexableArray<const TIndex> indices_iarray{indices};
        Indexer<> indices_indexer{indices.shape()};

        // size of (Ni..., Nj...) part
        TIndex common_total_size = gsl::narrow<TIndex>(a_indexer.total_size() / a_shape[0]);

        TIndex axis_dim = gsl::narrow<TIndex>(a_shape[0]);

        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&AddAtCudaKernel<T, TIndex>).block_size;
        int64_t total_size = out_indexer.total_size();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        AddAtCudaKernel<<<grid_size, block_size>>>(
                a_iarray, b_iarray, out_iarray, indices_iarray, b_indexer, out_indexer, indices_indexer, common_total_size, axis_dim);
    });
}

class CudaTakeKernel : public TakeKernel {
public:
    void Call(const Array& a, const Array& indices, int8_t axis, const Array& out) override {
        Device& device = a.device();
        CHAINERX_ASSERT(GetKind(indices.dtype()) == DtypeKind::kInt || GetKind(indices.dtype()) == DtypeKind::kUInt);
        device.CheckDevicesCompatible(a, indices, out);

        CudaSetDeviceScope scope{device.index()};

        if (indices.dtype() == Dtype::kInt64) {
            TakeImpl<int64_t>(device, a, indices, axis, out);
        } else {
            const Array& indices_cast = indices.dtype() == Dtype::kInt32 ? indices : indices.AsType(Dtype::kInt32);
            TakeImpl<int32_t>(device, a, indices_cast, axis, out);
        }
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(TakeKernel, CudaTakeKernel);

class CudaAddAtKernel : public AddAtKernel {
public:
    void Call(const Array& a, const Array& indices, int8_t axis, const Array& b, const Array& out) override {
        Device& device = a.device();
        CHAINERX_ASSERT(GetKind(indices.dtype()) == DtypeKind::kInt || GetKind(indices.dtype()) == DtypeKind::kUInt);
        device.CheckDevicesCompatible(a, indices, out);

        CudaSetDeviceScope scope{device.index()};

        if (indices.dtype() == Dtype::kInt64) {
            AddAtImpl<int64_t>(device, a, indices, axis, b, out);
        } else {
            const Array& indices_cast = indices.dtype() == Dtype::kInt32 ? indices : indices.AsType(Dtype::kInt32);
            AddAtImpl<int32_t>(device, a, indices_cast, axis, b, out);
        }
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(AddAtKernel, CudaAddAtKernel);

}  // namespace
}  // namespace cuda
}  // namespace chainerx
