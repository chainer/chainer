#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/arithmetic_ops.h"
#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/float16.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/kernels/math.h"
#include "chainerx/routines/math.h"
#include "chainerx/scalar.h"

namespace chainerx {
namespace cuda {
namespace {

CHAINERX_CUDA_REGISTER_ELTWISE_BINARY_KERNEL(AddKernel, { out = ArithmeticOps<CudaType>::Add(x1, x2); });

template <typename T>
struct AddASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Add(x1, x2); }
    CudaType x2;
};

class CudaAddASKernel : public AddASKernel {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(AddASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(AddASKernel, CudaAddASKernel);

CHAINERX_CUDA_REGISTER_ELTWISE_DTYPE_BINARY_KERNEL(SubtractKernel, { out = ArithmeticOps<CudaType>::Subtract(x1, x2); }, VisitNumericDtype);

template <typename T>
struct SubtractASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Subtract(x1, x2); }
    CudaType x2;
};

class CudaSubtractASKernel : public SubtractASKernel {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(SubtractASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(SubtractASKernel, CudaSubtractASKernel);

// TODO(sonots): support stream
CHAINERX_CUDA_REGISTER_ELTWISE_BINARY_KERNEL(MultiplyKernel, { out = ArithmeticOps<CudaType>::Multiply(x1, x2); });

template <typename T>
struct MultiplyASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Multiply(x1, x2); }
    CudaType x2;
};

class CudaMultiplyASKernel : public MultiplyASKernel {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(MultiplyASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(MultiplyASKernel, CudaMultiplyASKernel);

// CUDA does not have std::div.
__device__ int8_t FloorDivide(int8_t x, int8_t y) { return x / y - ((y >= 0 ? x % y : -(x % y)) < 0 ? 1 : 0); }
__device__ int16_t FloorDivide(int16_t x, int16_t y) { return x / y - ((y >= 0 ? x % y : -(x % y)) < 0 ? 1 : 0); }
__device__ int32_t FloorDivide(int32_t x, int32_t y) { return x / y - ((y >= 0 ? x % y : -(x % y)) < 0 ? 1 : 0); }
__device__ int64_t FloorDivide(int64_t x, int64_t y) { return x / y - ((y >= 0 ? x % y : -(x % y)) < 0 ? 1 : 0); }
__device__ uint8_t FloorDivide(uint8_t x, uint8_t y) { return x / y; }
__device__ float FloorDivide(float x, float y) {
    float rem = std::fmod(x, y);
    return (x - rem) / y - ((rem < 0 && y > 0) || (rem > 0 && y < 0) ? 1 : 0);
}
__device__ double FloorDivide(double x, double y) {
    double rem = std::fmod(x, y);
    return (x - rem) / y - ((rem < 0 && y > 0) || (rem > 0 && y < 0) ? 1 : 0);
}
__device__ cuda::Float16 FloorDivide(cuda::Float16 x, cuda::Float16 y) {
    return cuda::Float16{FloorDivide(static_cast<float>(x), static_cast<float>(y))};
}

CHAINERX_CUDA_REGISTER_ELTWISE_DTYPE_BINARY_KERNEL(FloorDivideKernel, { out = cuda::FloorDivide(x1, x2); }, VisitNumericDtype);

template <typename T>
struct FloorDivideASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = cuda::FloorDivide(x1, x2); }
    CudaType x2;
};

class CudaFloorDivideASKernel : public FloorDivideASKernel {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(FloorDivideASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(FloorDivideASKernel, CudaFloorDivideASKernel);

CHAINERX_CUDA_REGISTER_ELTWISE_BINARY_KERNEL(DivideKernel, { out = ArithmeticOps<CudaType>::Divide(x1, x2); });

template <typename T>
struct DivideASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Divide(x1, x2); }
    CudaType x2;
};

class CudaDivideASKernel : public DivideASKernel {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(DivideASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(DivideASKernel, CudaDivideASKernel);

}  // namespace
}  // namespace cuda
}  // namespace chainerx
