#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/arithmetic_ops.h"
#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/float16.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/scalar.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct AddImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, CudaType& out) { out = ArithmeticOps<CudaType>::Add(x1, x2); }
};

}  // namespace

// TODO(sonots): support stream
void CudaDevice::Add(const Array& x1, const Array& x2, const Array& out) {
    CheckDevicesCompatible(x1, x2, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, const T, T>(AddImpl<T>{}, x1, x2, out);
    });
}

namespace {

template <typename T>
struct AddASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Add(x1, x2); }
    CudaType x2;
};

}  // namespace

void CudaDevice::AddAS(const Array& x1, Scalar x2, const Array& out) {
    CheckDevicesCompatible(x1, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        using CudaType = cuda_internal::DataType<T>;
        Elementwise<const T, T>(AddASImpl<T>{static_cast<CudaType>(x2)}, x1, out);
    });
}

namespace {

template <typename T>
struct SubtractImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, CudaType& out) { out = ArithmeticOps<CudaType>::Subtract(x1, x2); }
};

}  // namespace

void CudaDevice::Subtract(const Array& x1, const Array& x2, const Array& out) {
    CheckDevicesCompatible(x1, x2, out);
    CudaSetDeviceScope scope{index()};
    VisitNumericDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, const T, T>(SubtractImpl<T>{}, x1, x2, out);
    });
}

namespace {

template <typename T>
struct SubtractASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Subtract(x1, x2); }
    CudaType x2;
};

}  // namespace

void CudaDevice::SubtractAS(const Array& x1, Scalar x2, const Array& out) {
    CheckDevicesCompatible(x1, out);
    CudaSetDeviceScope scope{index()};
    VisitNumericDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        using CudaType = cuda_internal::DataType<T>;
        Elementwise<const T, T>(SubtractASImpl<T>{static_cast<CudaType>(x2)}, x1, out);
    });
}

namespace {

template <typename T>
struct MultiplyImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, CudaType& out) { out = ArithmeticOps<CudaType>::Multiply(x1, x2); }
};

}  // namespace

// TODO(sonots): support stream
void CudaDevice::Multiply(const Array& x1, const Array& x2, const Array& out) {
    CheckDevicesCompatible(x1, x2, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, const T, T>(MultiplyImpl<T>{}, x1, x2, out);
    });
}

namespace {

template <typename T>
struct MultiplyASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Multiply(x1, x2); }
    CudaType x2;
};

}  // namespace

void CudaDevice::MultiplyAS(const Array& x1, Scalar x2, const Array& out) {
    CheckDevicesCompatible(x1, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        using CudaType = cuda_internal::DataType<T>;
        Elementwise<const T, T>(MultiplyASImpl<T>{static_cast<CudaType>(x2)}, x1, out);
    });
}

namespace {

// CUDA does not have std::div.
__device__ int8_t FloorDivide(int8_t x, int8_t y) { return x / y - ((y >= 0 ? x % y : -(x % y)) < 0 ? 1 : 0); }
__device__ int16_t FloorDivide(int16_t x, int16_t y) { return x / y - ((y >= 0 ? x % y : -(x % y)) < 0 ? 1 : 0); }
__device__ int32_t FloorDivide(int32_t x, int32_t y) { return x / y - ((y >= 0 ? x % y : -(x % y)) < 0 ? 1 : 0); }
__device__ int64_t FloorDivide(int64_t x, int64_t y) { return x / y - ((y >= 0 ? x % y : -(x % y)) < 0 ? 1 : 0); }
__device__ uint8_t FloorDivide(uint8_t x, uint8_t y) { return x / y; }
__device__ float FloorDivide(float x, float y) {
    float rem = std::fmod(x, y);
    return (x - rem) / y - ((rem < 0 && y > 0) || (rem > 0 && y < 0) ? 1 : 0);
}
__device__ double FloorDivide(double x, double y) {
    double rem = std::fmod(x, y);
    return (x - rem) / y - ((rem < 0 && y > 0) || (rem > 0 && y < 0) ? 1 : 0);
}
__device__ cuda::Float16 FloorDivide(cuda::Float16 x, cuda::Float16 y) {
    return cuda::Float16{FloorDivide(static_cast<float>(x), static_cast<float>(y))};
}

template <typename T>
struct FloorDivideImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, CudaType& out) { out = cuda::FloorDivide(x1, x2); }
};

}  // namespace

void CudaDevice::FloorDivide(const Array& x1, const Array& x2, const Array& out) {
    CheckDevicesCompatible(x1, x2, out);
    CudaSetDeviceScope scope{index()};
    VisitNumericDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, const T, T>(FloorDivideImpl<T>{}, x1, x2, out);
    });
}

namespace {

template <typename T>
struct FloorDivideASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = cuda::FloorDivide(x1, x2); }
    CudaType x2;
};

}  // namespace

void CudaDevice::FloorDivideAS(const Array& x1, Scalar x2, const Array& out) {
    CheckDevicesCompatible(x1, out);
    CudaSetDeviceScope scope{index()};
    VisitNumericDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        using CudaType = cuda_internal::DataType<T>;
        Elementwise<const T, T>(FloorDivideASImpl<T>{static_cast<CudaType>(x2)}, x1, out);
    });
}

namespace {

template <typename T>
struct DivideImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, CudaType& out) { out = ArithmeticOps<CudaType>::Divide(x1, x2); }
};

}  // namespace

void CudaDevice::Divide(const Array& x1, const Array& x2, const Array& out) {
    CheckDevicesCompatible(x1, x2, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, const T, T>(DivideImpl<T>{}, x1, x2, out);
    });
}

namespace {

template <typename T>
struct DivideASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Divide(x1, x2); }
    CudaType x2;
};

}  // namespace

void CudaDevice::DivideAS(const Array& x1, Scalar x2, const Array& out) {
    CheckDevicesCompatible(x1, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        using CudaType = cuda_internal::DataType<T>;
        Elementwise<const T, T>(DivideASImpl<T>{static_cast<CudaType>(x2)}, x1, out);
    });
}

}  // namespace cuda
}  // namespace chainerx
