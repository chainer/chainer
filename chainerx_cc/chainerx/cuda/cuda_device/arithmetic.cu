#include "chainerx/cuda/cuda_device.h"

#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/arithmetic_ops.h"
#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/scalar.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct AddImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, CudaType& out) { out = ArithmeticOps<CudaType>::Add(x1, x2); }
};

}  // namespace

// TODO(sonots): support stream
void CudaDevice::Add(const Array& x1, const Array& x2, const Array& out) {
    CheckDevicesCompatible(x1, x2, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, const T, T>(AddImpl<T>{}, x1, x2, out);
    });
}

namespace {

template <typename T>
struct AddASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Add(x1, x2); }
    CudaType x2;
};

}  // namespace

void CudaDevice::AddAS(const Array& x1, Scalar x2, const Array& out) {
    CheckDevicesCompatible(x1, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        using CudaType = cuda_internal::DataType<T>;
        Elementwise<const T, T>(AddASImpl<T>{static_cast<CudaType>(x2)}, x1, out);
    });
}

namespace {

template <typename T>
struct SubtractImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, CudaType& out) { out = ArithmeticOps<CudaType>::Subtract(x1, x2); }
};

}  // namespace

void CudaDevice::Subtract(const Array& x1, const Array& x2, const Array& out) {
    CheckDevicesCompatible(x1, x2, out);
    CudaSetDeviceScope scope{index()};
    VisitNumericDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, const T, T>(SubtractImpl<T>{}, x1, x2, out);
    });
}

namespace {

template <typename T>
struct SubtractASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Subtract(x1, x2); }
    CudaType x2;
};

}  // namespace

void CudaDevice::SubtractAS(const Array& x1, Scalar x2, const Array& out) {
    CheckDevicesCompatible(x1, out);
    CudaSetDeviceScope scope{index()};
    VisitNumericDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        using CudaType = cuda_internal::DataType<T>;
        Elementwise<const T, T>(SubtractASImpl<T>{static_cast<CudaType>(x2)}, x1, out);
    });
}

namespace {

template <typename T>
struct MultiplyImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, CudaType& out) { out = ArithmeticOps<CudaType>::Multiply(x1, x2); }
};

}  // namespace

// TODO(sonots): support stream
void CudaDevice::Multiply(const Array& x1, const Array& x2, const Array& out) {
    CheckDevicesCompatible(x1, x2, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, const T, T>(MultiplyImpl<T>{}, x1, x2, out);
    });
}

namespace {

template <typename T>
struct MultiplyASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Multiply(x1, x2); }
    CudaType x2;
};

}  // namespace

void CudaDevice::MultiplyAS(const Array& x1, Scalar x2, const Array& out) {
    CheckDevicesCompatible(x1, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        using CudaType = cuda_internal::DataType<T>;
        Elementwise<const T, T>(MultiplyASImpl<T>{static_cast<CudaType>(x2)}, x1, out);
    });
}

namespace {

template <typename T>
struct DivideImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, CudaType& out) { out = ArithmeticOps<CudaType>::Divide(x1, x2); }
};

}  // namespace

void CudaDevice::Divide(const Array& x1, const Array& x2, const Array& out) {
    CheckDevicesCompatible(x1, x2, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, const T, T>(DivideImpl<T>{}, x1, x2, out);
    });
}

namespace {

template <typename T>
struct DivideASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Divide(x1, x2); }
    CudaType x2;
};

}  // namespace

void CudaDevice::DivideAS(const Array& x1, Scalar x2, const Array& out) {
    CheckDevicesCompatible(x1, out);
    CudaSetDeviceScope scope{index()};
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        using CudaType = cuda_internal::DataType<T>;
        Elementwise<const T, T>(DivideASImpl<T>{static_cast<CudaType>(x2)}, x1, out);
    });
}

}  // namespace cuda
}  // namespace chainerx
