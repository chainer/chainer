#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/arithmetic_ops.h"
#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/float16.cuh"
#include "chainerx/cuda/op_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/routines/math.h"
#include "chainerx/scalar.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct AddImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, CudaType& out) { out = ArithmeticOps<CudaType>::Add(x1, x2); }
};

class CudaAddOp : public AddOp {
public:
    // TODO(sonots): support stream
    void Call(const Array& x1, const Array& x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, x2, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        const Array& x2_cast = x2.dtype() == out.dtype() ? x2 : x2.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, const T, T>(AddImpl<T>{}, x1_cast, x2_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(AddOp, CudaAddOp);

template <typename T>
struct AddASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Add(x1, x2); }
    CudaType x2;
};

class CudaAddASOp : public AddASOp {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(AddASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(AddASOp, CudaAddASOp);

template <typename T>
struct SubtractImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, CudaType& out) { out = ArithmeticOps<CudaType>::Subtract(x1, x2); }
};

class CudaSubtractOp : public SubtractOp {
public:
    void Call(const Array& x1, const Array& x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, x2, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        const Array& x2_cast = x2.dtype() == out.dtype() ? x2 : x2.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, const T, T>(SubtractImpl<T>{}, x1_cast, x2_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(SubtractOp, CudaSubtractOp);

template <typename T>
struct SubtractASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Subtract(x1, x2); }
    CudaType x2;
};

class CudaSubtractASOp : public SubtractASOp {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(SubtractASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(SubtractASOp, CudaSubtractASOp);

template <typename T>
struct MultiplyImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, CudaType& out) { out = ArithmeticOps<CudaType>::Multiply(x1, x2); }
};

// TODO(sonots): support stream
class CudaMultiplyOp : public MultiplyOp {
public:
    void Call(const Array& x1, const Array& x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, x2, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        const Array& x2_cast = x2.dtype() == out.dtype() ? x2 : x2.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, const T, T>(MultiplyImpl<T>{}, x1_cast, x2_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(MultiplyOp, CudaMultiplyOp);

template <typename T>
struct MultiplyASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Multiply(x1, x2); }
    CudaType x2;
};

class CudaMultiplyASOp : public MultiplyASOp {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(MultiplyASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(MultiplyASOp, CudaMultiplyASOp);

// CUDA does not have std::div.
__device__ int8_t FloorDivide(int8_t x, int8_t y) { return x / y - ((y >= 0 ? x % y : -(x % y)) < 0 ? 1 : 0); }
__device__ int16_t FloorDivide(int16_t x, int16_t y) { return x / y - ((y >= 0 ? x % y : -(x % y)) < 0 ? 1 : 0); }
__device__ int32_t FloorDivide(int32_t x, int32_t y) { return x / y - ((y >= 0 ? x % y : -(x % y)) < 0 ? 1 : 0); }
__device__ int64_t FloorDivide(int64_t x, int64_t y) { return x / y - ((y >= 0 ? x % y : -(x % y)) < 0 ? 1 : 0); }
__device__ uint8_t FloorDivide(uint8_t x, uint8_t y) { return x / y; }
__device__ float FloorDivide(float x, float y) {
    float rem = std::fmod(x, y);
    return (x - rem) / y - ((rem < 0 && y > 0) || (rem > 0 && y < 0) ? 1 : 0);
}
__device__ double FloorDivide(double x, double y) {
    double rem = std::fmod(x, y);
    return (x - rem) / y - ((rem < 0 && y > 0) || (rem > 0 && y < 0) ? 1 : 0);
}
__device__ cuda::Float16 FloorDivide(cuda::Float16 x, cuda::Float16 y) {
    return cuda::Float16{FloorDivide(static_cast<float>(x), static_cast<float>(y))};
}

template <typename T>
struct FloorDivideImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, CudaType& out) { out = cuda::FloorDivide(x1, x2); }
};

class CudaFloorDivideOp : public FloorDivideOp {
public:
    void Call(const Array& x1, const Array& x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, x2, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        const Array& x2_cast = x2.dtype() == out.dtype() ? x2 : x2.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, const T, T>(FloorDivideImpl<T>{}, x1_cast, x2_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(FloorDivideOp, CudaFloorDivideOp);

template <typename T>
struct FloorDivideASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = cuda::FloorDivide(x1, x2); }
    CudaType x2;
};

class CudaFloorDivideASOp : public FloorDivideASOp {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(FloorDivideASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(FloorDivideASOp, CudaFloorDivideASOp);

template <typename T>
struct DivideImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType x2, CudaType& out) { out = ArithmeticOps<CudaType>::Divide(x1, x2); }
};

class CudaDivideOp : public DivideOp {
public:
    void Call(const Array& x1, const Array& x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, x2, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        const Array& x2_cast = x2.dtype() == out.dtype() ? x2 : x2.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            Elementwise<const T, const T, T>(DivideImpl<T>{}, x1_cast, x2_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(DivideOp, CudaDivideOp);

template <typename T>
struct DivideASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Divide(x1, x2); }
    CudaType x2;
};

class CudaDivideASOp : public DivideASOp {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(DivideASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(DivideASOp, CudaDivideASOp);

}  // namespace
}  // namespace cuda
}  // namespace chainerx
