#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/arithmetic_ops.h"
#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/float16.cuh"
#include "chainerx/cuda/kernel_regist.h"
#include "chainerx/cuda/numeric.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/kernels/arithmetic.h"
#include "chainerx/routines/arithmetic.h"
#include "chainerx/scalar.h"

namespace chainerx {
namespace cuda {
namespace {

CHAINERX_CUDA_REGISTER_ELTWISE_BINARY_KERNEL(AddKernel, { out = ArithmeticOps<CudaType>::Add(x1, x2); });

template <typename T>
struct AddASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Add(x1, x2); }
    CudaType x2;
};

class CudaAddASKernel : public AddASKernel {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(AddASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(AddASKernel, CudaAddASKernel);

CHAINERX_CUDA_REGISTER_ELTWISE_DTYPE_BINARY_KERNEL(SubtractKernel, { out = ArithmeticOps<CudaType>::Subtract(x1, x2); }, VisitNumericDtype);

template <typename T>
struct SubtractASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Subtract(x1, x2); }
    CudaType x2;
};

class CudaSubtractASKernel : public SubtractASKernel {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(SubtractASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(SubtractASKernel, CudaSubtractASKernel);

// TODO(sonots): support stream
CHAINERX_CUDA_REGISTER_ELTWISE_BINARY_KERNEL(MultiplyKernel, { out = ArithmeticOps<CudaType>::Multiply(x1, x2); });

template <typename T>
struct MultiplyASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Multiply(x1, x2); }
    CudaType x2;
};

class CudaMultiplyASKernel : public MultiplyASKernel {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(MultiplyASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(MultiplyASKernel, CudaMultiplyASKernel);

// CUDA does not have std::div, which is used for the native backend.
template <typename T>
__device__ T FloorDivideImpl(T x, T y) {
    if (y == 0) {
        return 0;
    }
    return x / y - ((y >= 0 ? x % y : -(x % y)) < 0 ? 1 : 0);
}
__device__ int8_t FloorDivide(int8_t x, int8_t y) { return FloorDivideImpl(x, y); }
__device__ int16_t FloorDivide(int16_t x, int16_t y) { return FloorDivideImpl(x, y); }
__device__ int32_t FloorDivide(int32_t x, int32_t y) { return FloorDivideImpl(x, y); }
__device__ int64_t FloorDivide(int64_t x, int64_t y) { return FloorDivideImpl(x, y); }
__device__ uint8_t FloorDivide(uint8_t x, uint8_t y) {
    if (y == 0) {
        return 0;
    }
    return x / y;
}
__device__ float FloorDivide(float x, float y) {
    float rem = std::fmod(x, y);
    return (x - rem) / y - ((rem < 0 && y > 0) || (rem > 0 && y < 0) ? 1 : 0);
}
__device__ double FloorDivide(double x, double y) {
    double rem = std::fmod(x, y);
    return (x - rem) / y - ((rem < 0 && y > 0) || (rem > 0 && y < 0) ? 1 : 0);
}
__device__ cuda::Float16 FloorDivide(cuda::Float16 x, cuda::Float16 y) {
    return cuda::Float16{FloorDivide(static_cast<float>(x), static_cast<float>(y))};
}

CHAINERX_CUDA_REGISTER_ELTWISE_DTYPE_BINARY_KERNEL(FloorDivideKernel, { out = cuda::FloorDivide(x1, x2); }, VisitNumericDtype);

template <typename T>
struct FloorDivideASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = cuda::FloorDivide(x1, x2); }
    CudaType x2;
};

class CudaFloorDivideASKernel : public FloorDivideASKernel {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(FloorDivideASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(FloorDivideASKernel, CudaFloorDivideASKernel);

template <typename T>
struct FloorDivideSAImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x2, CudaType& out) { out = cuda::FloorDivide(x1, x2); }
    CudaType x1;
};

class CudaFloorDivideSAKernel : public FloorDivideSAKernel {
public:
    void Call(Scalar x1, const Array& x2, const Array& out) override {
        Device& device = x2.device();
        device.CheckDevicesCompatible(x2, out);
        const Array& x2_cast = x2.dtype() == out.dtype() ? x2 : x2.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(FloorDivideSAImpl<T>{static_cast<CudaType>(x1)}, x2_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(FloorDivideSAKernel, CudaFloorDivideSAKernel);

CHAINERX_CUDA_REGISTER_ELTWISE_BINARY_KERNEL(DivideKernel, { out = ArithmeticOps<CudaType>::Divide(x1, x2); });

template <typename T>
struct DivideASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = ArithmeticOps<CudaType>::Divide(x1, x2); }
    CudaType x2;
};

class CudaDivideASKernel : public DivideASKernel {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(DivideASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(DivideASKernel, CudaDivideASKernel);

template <typename T>
struct DivideSAImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x2, CudaType& out) { out = ArithmeticOps<CudaType>::Divide(x1, x2); }
    CudaType x1;
};

class CudaDivideSAKernel : public DivideSAKernel {
public:
    void Call(Scalar x1, const Array& x2, const Array& out) override {
        Device& device = x2.device();
        device.CheckDevicesCompatible(x2, out);
        const Array& x2_cast = x2.dtype() == out.dtype() ? x2 : x2.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(DivideSAImpl<T>{static_cast<CudaType>(x1)}, x2_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(DivideSAKernel, CudaDivideSAKernel);

CHAINERX_CUDA_REGISTER_ELTWISE_DTYPE_BINARY_KERNEL(PowerKernel, { out = cuda::Power(x1, x2); }, VisitNumericDtype);

template <typename T>
struct PowerASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = cuda::Power(x1, x2); }
    CudaType x2;
};

class CudaPowerASKernel : public PowerASKernel {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(PowerASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(PowerASKernel, CudaPowerASKernel);

template <typename T>
struct PowerSAImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x2, CudaType& out) { out = cuda::Power(x1, x2); }
    CudaType x1;
};

class CudaPowerSAKernel : public PowerSAKernel {
public:
    void Call(Scalar x1, const Array& x2, const Array& out) {
        Device& device = x2.device();
        device.CheckDevicesCompatible(x2, out);
        const Array& x2_cast = x2.dtype() == out.dtype() ? x2 : x2.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(PowerSAImpl<T>{static_cast<CudaType>(x1)}, x2_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(PowerSAKernel, CudaPowerSAKernel);

// CUDA does not have std::mod, which is used for the native backend.
template <typename T>
__device__ T ModSignedIntegerImpl(T x, T y) {
    if (x == 0 || y == 0) {
        return 0;
    }
    T ret = x % y;
    if ((ret > 0 && y < 0) || (ret < 0 && y > 0)) {
        return y + ret;
    }
    return ret;
}
__device__ int8_t Mod(int8_t x, int8_t y) { return ModSignedIntegerImpl(x, y); }
__device__ int16_t Mod(int16_t x, int16_t y) { return ModSignedIntegerImpl(x, y); }
__device__ int32_t Mod(int32_t x, int32_t y) { return ModSignedIntegerImpl(x, y); }
__device__ int64_t Mod(int64_t x, int64_t y) { return ModSignedIntegerImpl(x, y); }
__device__ uint8_t Mod(uint8_t x, uint8_t y) {
    if (x == 0 || y == 0) {
        return 0;
    }
    return x % y;
}
template <typename T>
__device__ T ModFloatImpl(T x, T y) {
    if (y == 0) {
        return NAN;
    }
    T ret = std::fmod(x, y);
    if ((ret > 0 && y < 0) || (ret < 0 && y > 0)) {
        return y + ret;
    }
    return ret;
}
__device__ double Mod(double x, double y) { return ModFloatImpl(x, y); }
__device__ float Mod(float x, float y) { return ModFloatImpl(x, y); }
__device__ cuda::Float16 Mod(cuda::Float16 x, cuda::Float16 y) { return cuda::Float16{Mod(static_cast<float>(x), static_cast<float>(y))}; }

CHAINERX_CUDA_REGISTER_ELTWISE_DTYPE_BINARY_KERNEL(ModAAKernel, { out = cuda::Mod(x1, x2); }, VisitNumericDtype);

template <typename T>
struct ModASImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x1, CudaType& out) { out = cuda::Mod(x1, x2); }
    CudaType x2;
};

class CudaModASKernel : public ModASKernel {
public:
    void Call(const Array& x1, Scalar x2, const Array& out) override {
        Device& device = x1.device();
        device.CheckDevicesCompatible(x1, out);
        const Array& x1_cast = x1.dtype() == out.dtype() ? x1 : x1.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(ModASImpl<T>{static_cast<CudaType>(x2)}, x1_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(ModASKernel, CudaModASKernel);

template <typename T>
struct ModSAImpl {
    using CudaType = cuda_internal::DataType<T>;
    __device__ void operator()(int64_t /*i*/, CudaType x2, CudaType& out) { out = cuda::Mod(x1, x2); }
    CudaType x1;
};

class CudaModSAKernel : public ModSAKernel {
public:
    void Call(Scalar x1, const Array& x2, const Array& out) override {
        Device& device = x2.device();
        device.CheckDevicesCompatible(x2, out);
        const Array& x2_cast = x2.dtype() == out.dtype() ? x2 : x2.AsType(out.dtype());
        CudaSetDeviceScope scope{device.index()};
        VisitNumericDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(ModSAImpl<T>{static_cast<CudaType>(x1)}, x2_cast, out);
        });
    }
};

CHAINERX_CUDA_REGISTER_KERNEL(ModSAKernel, CudaModSAKernel);

CHAINERX_CUDA_REGISTER_ELTWISE_BINARY_KERNEL(FmodKernel, { out = cuda::Fmod(x1, x2); });

}  // namespace
}  // namespace cuda
}  // namespace chainerx
